#include "hip/hip_runtime.h"
/*
Copyright (C) 2011 by Rio Yokota, Simon Layton, Lorena Barba

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
THE SOFTWARE.
*/
#define KERNEL
#include "kernel.h"
#undef KERNEL
#include "biotsavart.h"
#include "gpu.h"

template<>
void Kernel<BiotSavart>::initialize() {
  startTimer("Init GPU     ");                                  // Start timer
  hipDeviceReset();                                             // Exit GPU thread
  hipSetDevice(DEVICE);                                        // Set GPU device
  hipDeviceSynchronize();                                      // Sync GPU threads
  stopTimer("Init GPU     ",MPIRANK==0);                        // Stop timer & print
  eraseTimer("Init GPU     ");                                  // Erase timer
}

template<>
void Kernel<BiotSavart>::M2M_CPU(C_iter CI, C_iter CJ) {
  const complex I(0.,1.);                                   // Imaginary unit
  vect dist = CI->X - CJ->X;
  real rho, alpha, beta;
  cart2sph(rho,alpha,beta,dist);
  evalMultipole(rho,alpha,-beta);
  for( int j=0; j!=P; ++j ) {
    for( int k=0; k<=j; ++k ) {
      const int jk = j * j + j + k;
      const int jks = j * (j + 1) / 2 + k;
      complex M[3] = {0., 0., 0.};
      for( int n=0; n<=j; ++n ) {
        for( int m=-n; m<=std::min(k-1,n); ++m ) {
          if( j-n >= k-m ) {
            const int jnkm  = (j - n) * (j - n) + j - n + k - m;
            const int jnkms = (j - n) * (j - n + 1) / 2 + k - m;
            const int nm    = n * n + n + m;
            for( int d=0; d!=3; ++d ) {
              M[d] += CJ->M[3*jnkms+d] * std::pow(I,double(m-abs(m))) * Ynm[nm]
                    * double(ODDEVEN(n) * Anm[nm] * Anm[jnkm] / Anm[jk]);
            }
          }
        }
        for( int m=k; m<=n; ++m ) {
          if( j-n >= m-k ) {
            const int jnkm  = (j - n) * (j - n) + j - n + k - m;
            const int jnkms = (j - n) * (j - n + 1) / 2 - k + m;
            const int nm    = n * n + n + m;
            for( int d=0; d!=3; ++d ) {
              M[d] += std::conj(CJ->M[3*jnkms+d]) * Ynm[nm]
                    * double(ODDEVEN(k+n+m) * Anm[nm] * Anm[jnkm] / Anm[jk]);
            }
          }
        }
      }
      for( int d=0; d!=3; ++d ) {
        CI->M[3*jks+d] += M[d];
      }
    }
  }
}

template<>
void Kernel<BiotSavart>::finalize() {}

template<>
void Kernel<BiotSavart>::allocate() {
  hipDeviceSynchronize();
  startTimer("hipMalloc   ");
  if( keysHost.size() > keysDevcSize ) {
    if( keysDevcSize != 0 ) CUDA_SAFE_CALL(hipFree(keysDevc));
    CUDA_SAFE_CALL(hipMalloc( (void**) &keysDevc, keysHost.size()*sizeof(int) ));
    keysDevcSize = keysHost.size();
  }
  if( rangeHost.size() > rangeDevcSize ) {
    if( rangeDevcSize != 0 ) CUDA_SAFE_CALL(hipFree(rangeDevc));
    CUDA_SAFE_CALL(hipMalloc( (void**) &rangeDevc, rangeHost.size()*sizeof(int) ));
    rangeDevcSize = rangeHost.size();
  }
  if( sourceHost.size() > sourceDevcSize ) {
    if( sourceDevcSize != 0 ) CUDA_SAFE_CALL(hipFree(sourceDevc));
    CUDA_SAFE_CALL(hipMalloc( (void**) &sourceDevc, sourceHost.size()*sizeof(gpureal) ));
    sourceDevcSize = sourceHost.size();
  }
  if( targetHost.size() > targetDevcSize ) {
    if( targetDevcSize != 0 ) CUDA_SAFE_CALL(hipFree(targetDevc));
    CUDA_SAFE_CALL(hipMalloc( (void**) &targetDevc, targetHost.size()*sizeof(gpureal) ));
    targetDevcSize = targetHost.size();
  }
  hipDeviceSynchronize();
  stopTimer("hipMalloc   ");
}

template<>
void Kernel<BiotSavart>::hostToDevice() {
  hipDeviceSynchronize();
  startTimer("hipMemcpy   ");
  CUDA_SAFE_CALL(hipMemcpy(keysDevc,  &keysHost[0],  keysHost.size()*sizeof(int),hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(rangeDevc, &rangeHost[0], rangeHost.size()*sizeof(int),hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(sourceDevc,&sourceHost[0],sourceHost.size()*sizeof(gpureal),hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(targetDevc,&targetHost[0],targetHost.size()*sizeof(gpureal),hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constDevc),&constHost[0],constHost.size()*sizeof(gpureal)));
  hipDeviceSynchronize();
  stopTimer("hipMemcpy   ");
}

template<>
void Kernel<BiotSavart>::deviceToHost() {
  hipDeviceSynchronize();
  startTimer("hipMemcpy   ");
  CUDA_SAFE_CALL(hipMemcpy(&targetHost[0],targetDevc,targetHost.size()*sizeof(gpureal),hipMemcpyDeviceToHost));
  hipDeviceSynchronize();
  stopTimer("hipMemcpy   ");
}

__device__ void BiotSavartP2M_core(gpureal *target, gpureal rho, gpureal alpha, gpureal beta,
                                   gpureal *sourceShrd, int ithread) {
  __shared__ gpureal factShrd[2*P];
  gpureal Ynm;
  gpureal fact = 1;
  for( int i=0; i<2*P; ++i ) {
    factShrd[i] = fact;
    fact *= i + 1;
  }
  __syncthreads();
  int nn = floorf(sqrtf(2*threadIdx.x+0.25)-0.5);
  int mm = 0;
  for( int i=0; i<=nn; ++i ) mm += i;
  mm = threadIdx.x - mm;
  if( threadIdx.x >= NTERM ) nn = mm = 0;
  gpureal x = cosf(alpha);
  gpureal s = sqrtf(1 - x * x);
  fact = 1;
  gpureal pn = 1;
  gpureal rhom = 1;
  for( int m=0; m<mm; ++m ) {
    rhom *= rho;
    pn = -pn * fact * s;
    fact += 2;
  }
  int m=mm;
  gpureal p = pn;
  if(mm==nn) Ynm = rhom * p * rsqrtf(factShrd[2*m]);
  gpureal p1 = p;
  p = x * (2 * m + 1) * p;
  rhom *= rho;
  gpureal rhon = rhom;
  for( int n=m+1; n<=nn; ++n ) {
    if(n==nn){
      Ynm = rhon * p * rsqrtf(factShrd[n+m] / factShrd[n-m]);
    }
    gpureal p2 = p1;
    p1 = p;
    p = (x * (2 * n + 1) * p1 - (n + m) * p2) / (n - m + 1);
    rhon *= rho;
  }
  gpureal ere = cosf(-mm * beta);
  gpureal eim = sinf(-mm * beta);
  target[0] += sourceShrd[6*ithread+3] * Ynm * ere;
  target[1] += sourceShrd[6*ithread+3] * Ynm * eim;
  target[2] += sourceShrd[6*ithread+4] * Ynm * ere;
  target[3] += sourceShrd[6*ithread+4] * Ynm * eim;
  target[4] += sourceShrd[6*ithread+5] * Ynm * ere;
  target[5] += sourceShrd[6*ithread+5] * Ynm * eim;
}

__global__ void BiotSavartP2M_GPU(int *keysGlob, int *rangeGlob, gpureal *targetGlob, gpureal *sourceGlob) {
  int keys = keysGlob[blockIdx.x];
  int numList = rangeGlob[keys];
  gpureal target[6] = {0, 0, 0, 0, 0, 0};
  __shared__ gpureal targetShrd[3];
  __shared__ gpureal sourceShrd[6*THREADS];
  int itarget = blockIdx.x * THREADS;
  targetShrd[0] = targetGlob[6*itarget+0];
  targetShrd[1] = targetGlob[6*itarget+1];
  targetShrd[2] = targetGlob[6*itarget+2];
  for( int ilist=0; ilist<numList; ++ilist ) {
    int begin = rangeGlob[keys+3*ilist+1];
    int size  = rangeGlob[keys+3*ilist+2];
    for( int iblok=0; iblok<(size-1)/THREADS; ++iblok ) {
      int isource = begin + iblok * THREADS + threadIdx.x;
      __syncthreads();
      sourceShrd[6*threadIdx.x+0] = sourceGlob[7*isource+0];
      sourceShrd[6*threadIdx.x+1] = sourceGlob[7*isource+1];
      sourceShrd[6*threadIdx.x+2] = sourceGlob[7*isource+2];
      sourceShrd[6*threadIdx.x+3] = sourceGlob[7*isource+3];
      sourceShrd[6*threadIdx.x+4] = sourceGlob[7*isource+4];
      sourceShrd[6*threadIdx.x+5] = sourceGlob[7*isource+5];
      __syncthreads();
      for( int i=0; i<THREADS; ++i ) {
        float3 d;
        d.x = sourceShrd[6*i+0] - targetShrd[0];
        d.y = sourceShrd[6*i+1] - targetShrd[1];
        d.z = sourceShrd[6*i+2] - targetShrd[2];
        gpureal rho,alpha,beta;
        cart2sph(rho,alpha,beta,d.x,d.y,d.z);
        BiotSavartP2M_core(target,rho,alpha,beta,sourceShrd,i);
      }
    }
    int iblok = (size-1)/THREADS;
    int isource = begin + iblok * THREADS + threadIdx.x;
    __syncthreads();
    if( threadIdx.x < size - iblok * THREADS ) {
      sourceShrd[6*threadIdx.x+0] = sourceGlob[7*isource+0];
      sourceShrd[6*threadIdx.x+1] = sourceGlob[7*isource+1];
      sourceShrd[6*threadIdx.x+2] = sourceGlob[7*isource+2];
      sourceShrd[6*threadIdx.x+3] = sourceGlob[7*isource+3];
      sourceShrd[6*threadIdx.x+4] = sourceGlob[7*isource+4];
      sourceShrd[6*threadIdx.x+5] = sourceGlob[7*isource+5];
    }
    __syncthreads();
    for( int i=0; i<size-iblok*THREADS; ++i ) {
      float3 d;
      d.x = sourceShrd[6*i+0] - targetShrd[0];
      d.y = sourceShrd[6*i+1] - targetShrd[1];
      d.z = sourceShrd[6*i+2] - targetShrd[2];
      gpureal rho,alpha,beta;
      cart2sph(rho,alpha,beta,d.x,d.y,d.z);
      BiotSavartP2M_core(target,rho,alpha,beta,sourceShrd,i);
    }
  }
  itarget = blockIdx.x * THREADS + threadIdx.x;
  targetGlob[6*itarget+0] = target[0];
  targetGlob[6*itarget+1] = target[1];
  targetGlob[6*itarget+2] = target[2];
  targetGlob[6*itarget+3] = target[3];
  targetGlob[6*itarget+4] = target[4];
  targetGlob[6*itarget+5] = target[5];
}

template<>
void Kernel<BiotSavart>::P2M() {
  hipDeviceSynchronize();
  startTimer("P2M GPUkernel");
  int numBlocks = keysHost.size();\
  if( numBlocks != 0 ) {\
    BiotSavartP2M_GPU<<< numBlocks, THREADS >>>(keysDevc,rangeDevc,targetDevc,sourceDevc);\
  }\
  CUT_CHECK_ERROR("Kernel execution failed");\
  hipDeviceSynchronize();\
  stopTimer("P2M GPUkernel");\
}

__device__ void BiotSavartM2M_core(gpureal *target, gpureal beta, gpureal *factShrd, gpureal *YnmShrd, gpureal *sourceShrd) {
  int j = floorf(sqrtf(2*threadIdx.x+0.25)-0.5);
  int k = 0;
  for( int i=0; i<=j; ++i ) k += i;
  k = threadIdx.x - k;
  if( threadIdx.x >= NTERM ) j = k = 0;
  gpureal ajk = ODDEVEN(j) * rsqrtf(factShrd[j-k] * factShrd[j+k]);
  for( int n=0; n<=j; ++n ) {
    for( int m=-n; m<=min(k-1,n); ++m ) {
      if( j-n >= k-m ) {
        int nm = n * n + n + m;
        int jnkms = (j - n) * (j - n + 1) / 2 + k - m;
        gpureal ere = cosf(-m * beta);
        gpureal eim = sinf(-m * beta);
        gpureal ajnkm = rsqrtf(factShrd[j-n-k+m] * factShrd[j-n+k-m]);
        gpureal cnm = ODDEVEN((m-abs(m))/2+j);
        cnm *= ajnkm / ajk * YnmShrd[nm];
        gpureal CnmReal = cnm * ere;
        gpureal CnmImag = cnm * eim;
        target[0] += sourceShrd[6*jnkms+0] * CnmReal;
        target[0] -= sourceShrd[6*jnkms+1] * CnmImag;
        target[1] += sourceShrd[6*jnkms+0] * CnmImag;
        target[1] += sourceShrd[6*jnkms+1] * CnmReal;
        target[2] += sourceShrd[6*jnkms+2] * CnmReal;
        target[2] -= sourceShrd[6*jnkms+3] * CnmImag;
        target[3] += sourceShrd[6*jnkms+2] * CnmImag;
        target[3] += sourceShrd[6*jnkms+3] * CnmReal;
        target[4] += sourceShrd[6*jnkms+4] * CnmReal;
        target[4] -= sourceShrd[6*jnkms+5] * CnmImag;
        target[5] += sourceShrd[6*jnkms+4] * CnmImag;
        target[5] += sourceShrd[6*jnkms+5] * CnmReal;
      }
    }
    for( int m=k; m<=n; ++m ) {
      if( j-n >= m-k ) {
        int nm = n * n + n + m;
        int jnkms = (j - n) * (j - n + 1) / 2 - k + m;
        gpureal ere = cosf(-m * beta);
        gpureal eim = sinf(-m * beta);
        gpureal ajnkm = rsqrtf(factShrd[j-n-k+m] * factShrd[j-n+k-m]);
        gpureal cnm = ODDEVEN(k+j+m);
        cnm *= ajnkm / ajk * YnmShrd[nm];
        gpureal CnmReal = cnm * ere;
        gpureal CnmImag = cnm * eim;
        target[0] += sourceShrd[6*jnkms+0] * CnmReal;
        target[0] += sourceShrd[6*jnkms+1] * CnmImag;
        target[1] += sourceShrd[6*jnkms+0] * CnmImag;
        target[1] -= sourceShrd[6*jnkms+1] * CnmReal;
        target[2] += sourceShrd[6*jnkms+2] * CnmReal;
        target[2] += sourceShrd[6*jnkms+3] * CnmImag;
        target[3] += sourceShrd[6*jnkms+2] * CnmImag;
        target[3] -= sourceShrd[6*jnkms+3] * CnmReal;
        target[4] += sourceShrd[6*jnkms+4] * CnmReal;
        target[4] += sourceShrd[6*jnkms+5] * CnmImag;
        target[5] += sourceShrd[6*jnkms+4] * CnmImag;
        target[5] -= sourceShrd[6*jnkms+5] * CnmReal;
      }
    }
  }
}

__global__ void BiotSavartM2M_GPU(int *keysGlob, int *rangeGlob, gpureal *targetGlob, gpureal *sourceGlob) {
  int keys = keysGlob[blockIdx.x];
  int numList = rangeGlob[keys];
  gpureal target[6] = {0, 0, 0, 0, 0, 0};
  __shared__ gpureal sourceShrd[6*THREADS];
  __shared__ gpureal factShrd[2*P];
  __shared__ gpureal YnmShrd[P*P];
  gpureal fact = 1;
  for( int i=0; i<2*P; ++i ) {
    factShrd[i] = fact;
    fact *= i + 1;
  }
  __syncthreads();
  int itarget = blockIdx.x * THREADS;
  for( int ilist=0; ilist<numList; ++ilist ) {
    int begin = rangeGlob[keys+3*ilist+1];
    float3 d;
    d.x = targetGlob[6*itarget+0] - sourceGlob[begin+0];
    d.y = targetGlob[6*itarget+1] - sourceGlob[begin+1];
    d.z = targetGlob[6*itarget+2] - sourceGlob[begin+2];
    __syncthreads();
    if( threadIdx.x < NTERM ) {
      sourceShrd[6*threadIdx.x+0] = sourceGlob[begin+6*threadIdx.x+3];
      sourceShrd[6*threadIdx.x+1] = sourceGlob[begin+6*threadIdx.x+4];
      sourceShrd[6*threadIdx.x+2] = sourceGlob[begin+6*threadIdx.x+5];
      sourceShrd[6*threadIdx.x+3] = sourceGlob[begin+6*threadIdx.x+6];
      sourceShrd[6*threadIdx.x+4] = sourceGlob[begin+6*threadIdx.x+7];
      sourceShrd[6*threadIdx.x+5] = sourceGlob[begin+6*threadIdx.x+8];
    }
    __syncthreads();
    gpureal rho,alpha,beta;
    cart2sph(rho,alpha,beta,d.x,d.y,d.z);
    evalMultipole(YnmShrd,rho,alpha,factShrd);
    BiotSavartM2M_core(target,beta,factShrd,YnmShrd,sourceShrd);
  }
  itarget = blockIdx.x * THREADS + threadIdx.x;
  targetGlob[6*itarget+0] = target[0];
  targetGlob[6*itarget+1] = target[1];
  targetGlob[6*itarget+2] = target[2];
  targetGlob[6*itarget+3] = target[3];
  targetGlob[6*itarget+4] = target[4];
  targetGlob[6*itarget+5] = target[5];
}

template<>
void Kernel<BiotSavart>::M2M() {
  hipDeviceSynchronize();
  startTimer("M2M GPUkernel");
  int numBlocks = keysHost.size();\
  if( numBlocks != 0 ) {\
    BiotSavartM2M_GPU<<< numBlocks, THREADS >>>(keysDevc,rangeDevc,targetDevc,sourceDevc);\
  }\
  CUT_CHECK_ERROR("Kernel execution failed");\
  hipDeviceSynchronize();\
  stopTimer("M2M GPUkernel");\
}

__device__ void BiotSavartM2L_core(gpureal *target, gpureal  beta, gpureal *factShrd, gpureal *YnmShrd, gpureal *sourceShrd) {
  int j = floorf(sqrtf(2*threadIdx.x+0.25)-0.5);
  int k = 0;
  for( int i=0; i<=j; ++i ) k += i;
  k = threadIdx.x - k;
  if( threadIdx.x >= NTERM ) j = k = 0;
  gpureal ajk = ODDEVEN(j) * rsqrtf(factShrd[j-k] * factShrd[j+k]);
  for( int n=0; n<P; ++n ) {
    for( int m=-n; m<0; ++m ) {
      int jnkm = (j + n) * (j + n + 1) / 2 - m + k;
      gpureal ere = cosf((m - k) * beta);
      gpureal eim = sinf((m - k) * beta);
      gpureal anm = rsqrtf(factShrd[n-m] * factShrd[n+m]);
      gpureal cnm = anm * ajk * YnmShrd[jnkm];
      gpureal CnmReal = cnm * ere;
      gpureal CnmImag = cnm * eim;
      int i = n * (n + 1) / 2 - m;
      target[0] += sourceShrd[6*i+0] * CnmReal;
      target[0] += sourceShrd[6*i+1] * CnmImag;
      target[1] += sourceShrd[6*i+0] * CnmImag;
      target[1] -= sourceShrd[6*i+1] * CnmReal;
      target[2] += sourceShrd[6*i+2] * CnmReal;
      target[2] += sourceShrd[6*i+3] * CnmImag;
      target[3] += sourceShrd[6*i+2] * CnmImag;
      target[3] -= sourceShrd[6*i+3] * CnmReal;
      target[4] += sourceShrd[6*i+4] * CnmReal;
      target[4] += sourceShrd[6*i+5] * CnmImag;
      target[5] += sourceShrd[6*i+4] * CnmImag;
      target[5] -= sourceShrd[6*i+5] * CnmReal;
    }
    for( int m=0; m<=n; ++m ) {
      int jnkm = (j + n) * (j + n + 1) / 2 + abs(m - k);
      gpureal ere = cosf((m - k) * beta);
      gpureal eim = sinf((m - k) * beta);
      gpureal anm = rsqrtf(factShrd[n-m] * factShrd[n+m]);
      gpureal cnm = ODDEVEN((abs(k - m) - k - m) / 2);
      cnm *= anm * ajk * YnmShrd[jnkm];
      gpureal CnmReal = cnm * ere;
      gpureal CnmImag = cnm * eim;
      int i = n * (n + 1) / 2 + m;
      target[0] += sourceShrd[6*i+0] * CnmReal;
      target[0] -= sourceShrd[6*i+1] * CnmImag;
      target[1] += sourceShrd[6*i+0] * CnmImag;
      target[1] += sourceShrd[6*i+1] * CnmReal;
      target[2] += sourceShrd[6*i+2] * CnmReal;
      target[2] -= sourceShrd[6*i+3] * CnmImag;
      target[3] += sourceShrd[6*i+2] * CnmImag;
      target[3] += sourceShrd[6*i+3] * CnmReal;
      target[4] += sourceShrd[6*i+4] * CnmReal;
      target[4] -= sourceShrd[6*i+5] * CnmImag;
      target[5] += sourceShrd[6*i+4] * CnmImag;
      target[5] += sourceShrd[6*i+5] * CnmReal;
    }
  }
}

__global__ void BiotSavartM2L_GPU(int *keysGlob, int *rangeGlob, gpureal *targetGlob, gpureal *sourceGlob) {
  int keys = keysGlob[blockIdx.x];
  int numList = rangeGlob[keys];
  gpureal D0 = -constDevc[0];
  gpureal target[6] = {0, 0, 0, 0, 0, 0};
  __shared__ gpureal sourceShrd[6*THREADS];
  __shared__ gpureal factShrd[2*P];
  __shared__ gpureal YnmShrd[4*NTERM];
  gpureal fact = 1e-6;
  for( int i=0; i<2*P; ++i ) {
    factShrd[i] = fact;
    fact *= i + 1;
  }
  __syncthreads();
  int itarget = blockIdx.x * THREADS;
  for( int ilist=0; ilist<numList; ++ilist ) {
    int begin     = rangeGlob[keys+3*ilist+1];
    int Iperiodic = rangeGlob[keys+3*ilist+3];
    __syncthreads();
    if( threadIdx.x < NTERM ) {
      sourceShrd[6*threadIdx.x+0] = sourceGlob[begin+6*threadIdx.x+3];
      sourceShrd[6*threadIdx.x+1] = sourceGlob[begin+6*threadIdx.x+4];
      sourceShrd[6*threadIdx.x+2] = sourceGlob[begin+6*threadIdx.x+5];
      sourceShrd[6*threadIdx.x+3] = sourceGlob[begin+6*threadIdx.x+6];
      sourceShrd[6*threadIdx.x+4] = sourceGlob[begin+6*threadIdx.x+7];
      sourceShrd[6*threadIdx.x+5] = sourceGlob[begin+6*threadIdx.x+8];
    }
    __syncthreads();
    int I = 0;
    for( int ix=-1; ix<=1; ++ix ) {
      for( int iy=-1; iy<=1; ++iy ) {
        for( int iz=-1; iz<=1; ++iz, ++I ) {
          if( Iperiodic & (1 << I) ) {
            float3 d;
            d.x = ix * D0;
            d.y = iy * D0;
            d.z = iz * D0;
            d.x += targetGlob[6*itarget+0] - sourceGlob[begin+0];
            d.y += targetGlob[6*itarget+1] - sourceGlob[begin+1];
            d.z += targetGlob[6*itarget+2] - sourceGlob[begin+2];
            gpureal rho,alpha,beta;
            cart2sph(rho,alpha,beta,d.x,d.y,d.z);
            evalLocal(YnmShrd,rho,alpha,factShrd);
            BiotSavartM2L_core(target,beta,factShrd,YnmShrd,sourceShrd);
          }
        }
      }
    }
  }
  itarget = blockIdx.x * THREADS + threadIdx.x;
  targetGlob[6*itarget+0] = target[0] * 1e-6;
  targetGlob[6*itarget+1] = target[1] * 1e-6;
  targetGlob[6*itarget+2] = target[2] * 1e-6;
  targetGlob[6*itarget+3] = target[3] * 1e-6;
  targetGlob[6*itarget+4] = target[4] * 1e-6;
  targetGlob[6*itarget+5] = target[5] * 1e-6;
}

template<>
void Kernel<BiotSavart>::M2L() {
  hipDeviceSynchronize();
  startTimer("M2L GPUkernel");
  int numBlocks = keysHost.size();\
  if( numBlocks != 0 ) {\
    BiotSavartM2L_GPU<<< numBlocks, THREADS >>>(keysDevc,rangeDevc,targetDevc,sourceDevc);\
  }\
  CUT_CHECK_ERROR("Kernel execution failed");\
  hipDeviceSynchronize();\
  stopTimer("M2L GPUkernel");\
}

__device__ void BiotSavartM2P_core(gpureal *target, gpureal r, gpureal theta, gpureal phi, gpureal *factShrd, gpureal *sourceShrd) {
  gpureal x = cosf(theta);
  gpureal y = sinf(theta);
  if( fabsf(y) < EPS ) y = 1 / EPS;
  gpureal s = sqrtf(1 - x * x);
  gpureal spherical[9] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
  gpureal cartesian[9] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
  gpureal fact = 1;
  gpureal pn = 1;
  gpureal rhom = 1 / r;
  for( int m=0; m<P; ++m ) {
    gpureal p = pn;
    int i = m * (m + 1) / 2 + m;
    gpureal ere = cosf(m * phi);
    if( m == 0 ) ere = 0.5;
    gpureal eim = sinf(m * phi);
    gpureal anm = rhom * rsqrtf(factShrd[2*m]);
    gpureal Ynm = anm * p;
    gpureal p1 = p;
    p = x * (2 * m + 1) * p;
    gpureal YnmTheta = anm * (p - (m + 1) * x * p1) / y;
    gpureal realj = ere * sourceShrd[6*i+0] - eim * sourceShrd[6*i+1];
    gpureal imagj = eim * sourceShrd[6*i+0] + ere * sourceShrd[6*i+1];
    spherical[0] -= 2 * (m + 1) / r * Ynm * realj;
    spherical[1] += 2 * YnmTheta * realj;
    spherical[2] -= 2 * m * Ynm * imagj;
    realj = ere * sourceShrd[6*i+2] - eim * sourceShrd[6*i+3];
    imagj = eim * sourceShrd[6*i+2] + ere * sourceShrd[6*i+3];
    spherical[3] -= 2 * (m + 1) / r * Ynm * realj;
    spherical[4] += 2 * YnmTheta * realj;
    spherical[5] -= 2 * m * Ynm * imagj;
    realj = ere * sourceShrd[6*i+4] - eim * sourceShrd[6*i+5];
    imagj = eim * sourceShrd[6*i+4] + ere * sourceShrd[6*i+5];
    spherical[6] -= 2 * (m + 1) / r * Ynm * realj;
    spherical[7] += 2 * YnmTheta * realj;
    spherical[8] -= 2 * m * Ynm * imagj;
    rhom /= r;
    gpureal rhon = rhom;
    for( int n=m+1; n<P; ++n ) {
      i = n * (n + 1) / 2 + m;
      anm = rhon * rsqrtf(factShrd[n+m] / factShrd[n-m]);
      Ynm = anm * p;
      gpureal p2 = p1;
      p1 = p;
      p = (x * (2 * n + 1) * p1 - (n + m) * p2) / (n - m + 1);
      YnmTheta = anm * ((n - m + 1) * p - (n + 1) * x * p1) / y;
      realj = ere * sourceShrd[6*i+0] - eim * sourceShrd[6*i+1];
      imagj = eim * sourceShrd[6*i+0] + ere * sourceShrd[6*i+1];
      spherical[0] -= 2 * (n + 1) / r * Ynm * realj;
      spherical[1] += 2 * YnmTheta * realj;
      spherical[2] -= 2 * m * Ynm * imagj;
      realj = ere * sourceShrd[6*i+2] - eim * sourceShrd[6*i+3];
      imagj = eim * sourceShrd[6*i+2] + ere * sourceShrd[6*i+3];
      spherical[3] -= 2 * (n + 1) / r * Ynm * realj;
      spherical[4] += 2 * YnmTheta * realj;
      spherical[5] -= 2 * m * Ynm * imagj;
      realj = ere * sourceShrd[6*i+4] - eim * sourceShrd[6*i+5];
      imagj = eim * sourceShrd[6*i+4] + ere * sourceShrd[6*i+5];
      spherical[6] -= 2 * (n + 1) / r * Ynm * realj;
      spherical[7] += 2 * YnmTheta * realj;
      spherical[8] -= 2 * m * Ynm * imagj;
      rhon /= r;
    }
    pn = -pn * fact * s;
    fact += 2;
  }
  sph2cart(r,theta,phi,&spherical[0],&cartesian[0]);
  sph2cart(r,theta,phi,&spherical[3],&cartesian[3]);
  sph2cart(r,theta,phi,&spherical[6],&cartesian[6]);
  target[0] += 0.25 / M_PI * (cartesian[5] - cartesian[7]);
  target[1] += 0.25 / M_PI * (cartesian[6] - cartesian[2]);
  target[2] += 0.25 / M_PI * (cartesian[1] - cartesian[3]);
}

__global__ void BiotSavartM2P_GPU(int *keysGlob, int *rangeGlob, gpureal *targetGlob, gpureal *sourceGlob) {
  int keys = keysGlob[blockIdx.x];
  int numList = rangeGlob[keys];
  gpureal D0 = -constDevc[0];
  gpureal targetX[3];
  gpureal target[3] = {0, 0, 0};
  __shared__ gpureal sourceShrd[6*THREADS];
  __shared__ gpureal factShrd[2*P];
  gpureal fact = 1;
  for( int i=0; i<2*P; ++i ) {
    factShrd[i] = fact;
    fact *= i + 1;
  }
  __syncthreads();
  int itarget = blockIdx.x * THREADS + threadIdx.x;
  targetX[0] = targetGlob[6*itarget+0];
  targetX[1] = targetGlob[6*itarget+1];
  targetX[2] = targetGlob[6*itarget+2];
  for( int ilist=0; ilist<numList; ++ilist ) {
    int begin     = rangeGlob[keys+3*ilist+1];
    int Iperiodic = rangeGlob[keys+3*ilist+3];
    __syncthreads();
    if( threadIdx.x < NTERM ) {
      sourceShrd[6*threadIdx.x+0] = sourceGlob[begin+6*threadIdx.x+3];
      sourceShrd[6*threadIdx.x+1] = sourceGlob[begin+6*threadIdx.x+4];
      sourceShrd[6*threadIdx.x+2] = sourceGlob[begin+6*threadIdx.x+5];
      sourceShrd[6*threadIdx.x+3] = sourceGlob[begin+6*threadIdx.x+6];
      sourceShrd[6*threadIdx.x+4] = sourceGlob[begin+6*threadIdx.x+7];
      sourceShrd[6*threadIdx.x+5] = sourceGlob[begin+6*threadIdx.x+8];
    }
    __syncthreads();
    int I = 0;
    for( int ix=-1; ix<=1; ++ix ) {
      for( int iy=-1; iy<=1; ++iy ) {
        for( int iz=-1; iz<=1; ++iz, ++I ) {
          if( Iperiodic & (1 << I) ) {
            float3 d;
            d.x = ix * D0;
            d.y = iy * D0;
            d.z = iz * D0;
            d.x += targetX[0] - sourceGlob[begin+0];
            d.y += targetX[1] - sourceGlob[begin+1];
            d.z += targetX[2] - sourceGlob[begin+2];
            gpureal r,theta,phi;
            cart2sph(r,theta,phi,d.x,d.y,d.z);
            BiotSavartM2P_core(target,r,theta,phi,factShrd,sourceShrd);
          }
        }
      }
    }
  }
  targetGlob[6*itarget+0] = target[0];
  targetGlob[6*itarget+1] = target[1];
  targetGlob[6*itarget+2] = target[2];
}

template<>
void Kernel<BiotSavart>::M2P() {
  hipDeviceSynchronize();
  startTimer("M2P GPUkernel");
  int numBlocks = keysHost.size();\
  if( numBlocks != 0 ) {\
    BiotSavartM2P_GPU<<< numBlocks, THREADS >>>(keysDevc,rangeDevc,targetDevc,sourceDevc);\
  }\
  CUT_CHECK_ERROR("Kernel execution failed");\
  hipDeviceSynchronize();\
  stopTimer("M2P GPUkernel");\
}

__device__ inline void BiotSavartP2P_core(gpureal *target, gpureal *targetX, gpureal *sourceShrd, float3 d, int i) {
  d.x += targetX[0];
  d.x -= sourceShrd[7*i+0];
  d.y += targetX[1];
  d.y -= sourceShrd[7*i+1];
  d.z += targetX[2];
  d.z -= sourceShrd[7*i+2];
#if 0
  gpureal R2 = d.x * d.x + d.y * d.y + d.z * d.z + EPS2;
  gpureal RS = R2 * sourceShrd[7*i+6];
  gpureal invR = rsqrtf(R2);
  if( R2 == 0 ) invR = 0;
  gpureal invR3 = invR * invR * invR;
  gpureal cutoff = 0.25 / M_PI * invR3 * (erff( sqrtf(RS) )
               - sqrtf(4 / M_PI * RS) * expf(-RS));
  target[0] += (d.y * sourceShrd[7*i+5] - d.z * sourceShrd[7*i+4]) * cutoff;
  target[1] += (d.z * sourceShrd[7*i+3] - d.x * sourceShrd[7*i+5]) * cutoff;
  target[2] += (d.x * sourceShrd[7*i+4] - d.y * sourceShrd[7*i+3]) * cutoff;
#else
  const gpureal SQRT4PI = M_2_SQRTPI;
  const gpureal FOURPI = 0.25 * M_1_PI;
  gpureal R2 = d.x * d.x + d.y * d.y + d.z * d.z + EPS2;
  gpureal invR = rsqrtf(R2);
  if( R2 == 0 ) invR = 0;
  gpureal invR3 = invR * invR * invR;
  gpureal RS = R2 * sourceShrd[7*i+6];
  gpureal SQRT_RS = sqrtf(RS);
  gpureal z = SQRT_RS,t,r;
  (t)=1.0f/(1.0f+0.5f*(z));
  (r)=(t)*expf(-(z)*(z)-1.26551223f+(t)*(1.00002368f+(t)*(0.37409196f+(t)*(0.09678418f+
      (t)*(-0.18628806f+(t)*(0.27886807f+(t)*(-1.13520398f+(t)*(1.48851587f+
      (t)*(-0.82215223f+(t)*0.17087277f)))))))));
  gpureal cutoff = FOURPI * invR3 * ( 1.0f - r
               - SQRT4PI * SQRT_RS * expf(-RS));
  target[0] += (d.y * sourceShrd[7*i+5] - d.z * sourceShrd[7*i+4]) * cutoff;
  target[1] += (d.z * sourceShrd[7*i+3] - d.x * sourceShrd[7*i+5]) * cutoff;
  target[2] += (d.x * sourceShrd[7*i+4] - d.y * sourceShrd[7*i+3]) * cutoff;
#endif
}

__global__ void BiotSavartP2P_GPU(int *keysGlob, int *rangeGlob, gpureal *targetGlob, gpureal *sourceGlob) {
  int keys = keysGlob[blockIdx.x];
  int numList = rangeGlob[keys];
  gpureal D0 = -constDevc[0];
  gpureal targetX[3];
  gpureal target[3] = {0, 0, 0};
  __shared__ gpureal sourceShrd[7*THREADS];
  int itarget = blockIdx.x * THREADS + threadIdx.x;
  targetX[0] = targetGlob[6*itarget+0];
  targetX[1] = targetGlob[6*itarget+1];
  targetX[2] = targetGlob[6*itarget+2];
  for( int ilist=0; ilist<numList; ++ilist ) {
    int begin     = rangeGlob[keys+3*ilist+1];
    int size      = rangeGlob[keys+3*ilist+2];
    int Iperiodic = rangeGlob[keys+3*ilist+3];
    for( int iblok=0; iblok<(size-1)/THREADS; ++iblok ) {
      int isource = begin + iblok * THREADS + threadIdx.x;
      __syncthreads();
      sourceShrd[7*threadIdx.x+0] = sourceGlob[7*isource+0];
      sourceShrd[7*threadIdx.x+1] = sourceGlob[7*isource+1];
      sourceShrd[7*threadIdx.x+2] = sourceGlob[7*isource+2];
      sourceShrd[7*threadIdx.x+3] = sourceGlob[7*isource+3];
      sourceShrd[7*threadIdx.x+4] = sourceGlob[7*isource+4];
      sourceShrd[7*threadIdx.x+5] = sourceGlob[7*isource+5];
      sourceShrd[7*threadIdx.x+6] = 0.5f / (sourceGlob[7*isource+6] * sourceGlob[7*isource+6]);
      __syncthreads();
      int I = 0;
      for( int ix=-1; ix<=1; ++ix ) {
        for( int iy=-1; iy<=1; ++iy ) {
          for( int iz=-1; iz<=1; ++iz, ++I ) {
            if( Iperiodic & (1 << I) ) {
              float3 d;
              d.x = ix * D0;
              d.y = iy * D0;
              d.z = iz * D0;
#pragma unroll 64
              for( int i=0; i<THREADS; ++i ) {
                BiotSavartP2P_core(target,targetX,sourceShrd,d,i);
              }
            }
          }
        }
      }
    }
    int iblok = (size-1)/THREADS;
    int isource = begin + iblok * THREADS + threadIdx.x;
    __syncthreads();
    if( threadIdx.x < size - iblok * THREADS ) {
      sourceShrd[7*threadIdx.x+0] = sourceGlob[7*isource+0];
      sourceShrd[7*threadIdx.x+1] = sourceGlob[7*isource+1];
      sourceShrd[7*threadIdx.x+2] = sourceGlob[7*isource+2];
      sourceShrd[7*threadIdx.x+3] = sourceGlob[7*isource+3];
      sourceShrd[7*threadIdx.x+4] = sourceGlob[7*isource+4];
      sourceShrd[7*threadIdx.x+5] = sourceGlob[7*isource+5];
      sourceShrd[7*threadIdx.x+6] = 0.5f / (sourceGlob[7*isource+6] * sourceGlob[7*isource+6]);
    }
    __syncthreads();
    int I = 0;
    int icounter=0;
    for( int ix=-1; ix<=1; ++ix ) {
      for( int iy=-1; iy<=1; ++iy ) {
        for( int iz=-1; iz<=1; ++iz, ++I ) {
          if( Iperiodic & (1 << I) ) {
            icounter++;
            float3 d;
            d.x = ix * D0;
            d.y = iy * D0;
            d.z = iz * D0;
            for( int i=0; i<size-iblok*THREADS; ++i ) {
              BiotSavartP2P_core(target,targetX,sourceShrd,d,i);
            }
          }
        }
      }
    }
  }
  targetGlob[6*itarget+0] = target[0];
  targetGlob[6*itarget+1] = target[1];
  targetGlob[6*itarget+2] = target[2];
}

template<>
void Kernel<BiotSavart>::P2P() {
  hipDeviceSynchronize();
  startTimer("P2P GPUkernel");
  int numBlocks = keysHost.size();\
  if( numBlocks != 0 ) {\
    BiotSavartP2P_GPU<<< numBlocks, THREADS >>>(keysDevc,rangeDevc,targetDevc,sourceDevc);\
  }\
  CUT_CHECK_ERROR("Kernel execution failed");\
  hipDeviceSynchronize();\
  stopTimer("P2P GPUkernel");\
}

__device__ void BiotSavartL2L_core(gpureal *target, gpureal beta, gpureal *factShrd, gpureal *YnmShrd, gpureal *sourceShrd) {
  int j = floorf(sqrtf(2*threadIdx.x+0.25)-0.5);
  int k = 0;
  for( int i=0; i<=j; ++i ) k += i;
  k = threadIdx.x - k;
  if( threadIdx.x >= NTERM ) j = k = 0;
  gpureal ajk = ODDEVEN(j) * rsqrtf(factShrd[j-k] * factShrd[j+k]);
  for( int n=0; n<P; ++n ) {
    for( int m=j+k-n; m<0; ++m ) {
      int nms = n * (n + 1) / 2 - m;
      int jnkm = (n - j) * (n - j) + n - j + m - k;
      gpureal ere = cosf((m - k) * beta);
      gpureal eim = sinf((m - k) * beta);
      gpureal anm = rsqrtf(factShrd[n-m] * factShrd[n+m]);
      gpureal cnm = ODDEVEN(k-n) * ajk / anm * YnmShrd[jnkm];
      gpureal CnmReal = cnm * ere;
      gpureal CnmImag = cnm * eim;
      target[0] += sourceShrd[6*nms+0] * CnmReal;
      target[0] += sourceShrd[6*nms+1] * CnmImag;
      target[1] += sourceShrd[6*nms+0] * CnmImag;
      target[1] -= sourceShrd[6*nms+1] * CnmReal;
      target[2] += sourceShrd[6*nms+2] * CnmReal;
      target[2] += sourceShrd[6*nms+3] * CnmImag;
      target[3] += sourceShrd[6*nms+2] * CnmImag;
      target[3] -= sourceShrd[6*nms+3] * CnmReal;
      target[4] += sourceShrd[6*nms+4] * CnmReal;
      target[4] += sourceShrd[6*nms+5] * CnmImag;
      target[5] += sourceShrd[6*nms+4] * CnmImag;
      target[5] -= sourceShrd[6*nms+5] * CnmReal;
    }
    for( int m=0; m<=n; ++m ) {
      if( n-j >= abs(m-k) ) {
        int nms = n * (n + 1) / 2 + m;
        int jnkm = (n - j) * (n - j) + n - j + m - k;
        gpureal ere = cosf((m - k) * beta);
        gpureal eim = sinf((m - k) * beta);
        gpureal anm = rsqrtf(factShrd[n-m] * factShrd[n+m]);
        gpureal cnm = ODDEVEN((m-k-abs(m-k)) / 2 - n);
        cnm *= ajk / anm * YnmShrd[jnkm];
        gpureal CnmReal = cnm * ere;
        gpureal CnmImag = cnm * eim;
        target[0] += sourceShrd[6*nms+0] * CnmReal;
        target[0] -= sourceShrd[6*nms+1] * CnmImag;
        target[1] += sourceShrd[6*nms+0] * CnmImag;
        target[1] += sourceShrd[6*nms+1] * CnmReal;
        target[2] += sourceShrd[6*nms+2] * CnmReal;
        target[2] -= sourceShrd[6*nms+3] * CnmImag;
        target[3] += sourceShrd[6*nms+2] * CnmImag;
        target[3] += sourceShrd[6*nms+3] * CnmReal;
        target[4] += sourceShrd[6*nms+4] * CnmReal;
        target[4] -= sourceShrd[6*nms+5] * CnmImag;
        target[5] += sourceShrd[6*nms+4] * CnmImag;
        target[5] += sourceShrd[6*nms+5] * CnmReal;
      }
    }
  }
}

__global__ void BiotSavartL2L_GPU(int *keysGlob, int *rangeGlob, gpureal *targetGlob, gpureal *sourceGlob) {
  int keys = keysGlob[blockIdx.x];
  int numList = rangeGlob[keys];
  gpureal target[6] = {0, 0, 0, 0, 0, 0};
  __shared__ gpureal sourceShrd[6*THREADS];
  __shared__ gpureal factShrd[2*P];
  __shared__ gpureal YnmShrd[P*P];
  gpureal fact = 1;
  for( int i=0; i<2*P; ++i ) {
    factShrd[i] = fact;
    fact *= i + 1;
  }
  __syncthreads();
  int itarget = blockIdx.x * THREADS;
  for( int ilist=0; ilist<numList; ++ilist ) {
    int begin = rangeGlob[keys+3*ilist+1];
    float3 d;
    d.x = targetGlob[6*itarget+0] - sourceGlob[begin+0];
    d.y = targetGlob[6*itarget+1] - sourceGlob[begin+1];
    d.z = targetGlob[6*itarget+2] - sourceGlob[begin+2];
    __syncthreads();
    if( threadIdx.x < NTERM ) {
      sourceShrd[6*threadIdx.x+0] = sourceGlob[begin+6*threadIdx.x+3];
      sourceShrd[6*threadIdx.x+1] = sourceGlob[begin+6*threadIdx.x+4];
      sourceShrd[6*threadIdx.x+2] = sourceGlob[begin+6*threadIdx.x+5];
      sourceShrd[6*threadIdx.x+3] = sourceGlob[begin+6*threadIdx.x+6];
      sourceShrd[6*threadIdx.x+4] = sourceGlob[begin+6*threadIdx.x+7];
      sourceShrd[6*threadIdx.x+5] = sourceGlob[begin+6*threadIdx.x+8];
    }
    __syncthreads();
    gpureal rho,alpha,beta;
    cart2sph(rho,alpha,beta,d.x,d.y,d.z);
    evalMultipole(YnmShrd,rho,alpha,factShrd);
    BiotSavartL2L_core(target,beta,factShrd,YnmShrd,sourceShrd);
  }
  itarget = blockIdx.x * THREADS + threadIdx.x;
  targetGlob[6*itarget+0] = target[0];
  targetGlob[6*itarget+1] = target[1];
  targetGlob[6*itarget+2] = target[2];
  targetGlob[6*itarget+3] = target[3];
  targetGlob[6*itarget+4] = target[4];
  targetGlob[6*itarget+5] = target[5];
}

template<>
void Kernel<BiotSavart>::L2L() {
  hipDeviceSynchronize();
  startTimer("L2L GPUkernel");
  int numBlocks = keysHost.size();\
  if( numBlocks != 0 ) {\
    BiotSavartL2L_GPU<<< numBlocks, THREADS >>>(keysDevc,rangeDevc,targetDevc,sourceDevc);\
  }\
  CUT_CHECK_ERROR("Kernel execution failed");\
  hipDeviceSynchronize();\
  stopTimer("L2L GPUkernel");\
}

__device__ void BiotSavartL2P_core(gpureal *target, gpureal r, gpureal theta, gpureal phi, gpureal *factShrd, gpureal *sourceShrd) {
  gpureal x = cosf(theta);
  gpureal y = sinf(theta);
  if( fabsf(y) < EPS ) y = 1 / EPS;
  gpureal s = sqrtf(1 - x * x);
  gpureal spherical[9] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
  gpureal cartesian[9] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
  gpureal fact = 1;
  gpureal pn = 1;
  gpureal rhom = 1;
  for( int m=0; m<P; ++m ) {
    gpureal p = pn;
    int i = m * (m + 1) / 2 + m;
    gpureal ere = cosf(m * phi);
    if( m == 0 ) ere = 0.5;
    gpureal eim = sinf(m * phi);
    gpureal anm = rhom * rsqrtf(factShrd[2*m]);
    gpureal Ynm = anm * p;
    gpureal p1 = p;
    p = x * (2 * m + 1) * p;
    gpureal YnmTheta = anm * (p - (m + 1) * x * p1) / y;
    gpureal realj = ere * sourceShrd[6*i+0] - eim * sourceShrd[6*i+1];
    gpureal imagj = eim * sourceShrd[6*i+0] + ere * sourceShrd[6*i+1];
    spherical[0] += 2 * m / r * Ynm * realj;
    spherical[1] += 2 * YnmTheta * realj;
    spherical[2] -= 2 * m * Ynm * imagj;
    realj = ere * sourceShrd[6*i+2] - eim * sourceShrd[6*i+3];
    imagj = eim * sourceShrd[6*i+2] + ere * sourceShrd[6*i+3];
    spherical[3] += 2 * m / r * Ynm * realj;
    spherical[4] += 2 * YnmTheta * realj;
    spherical[5] -= 2 * m * Ynm * imagj;
    realj = ere * sourceShrd[6*i+4] - eim * sourceShrd[6*i+5];
    imagj = eim * sourceShrd[6*i+4] + ere * sourceShrd[6*i+5];
    spherical[6] += 2 * m / r * Ynm * realj;
    spherical[7] += 2 * YnmTheta * realj;
    spherical[8] -= 2 * m * Ynm * imagj;
    rhom *= r;
    gpureal rhon = rhom;
    for( int n=m+1; n<P; ++n ) {
      i = n * (n + 1) / 2 + m;
      anm = rhon * rsqrtf(factShrd[n+m] / factShrd[n-m]);
      Ynm = anm * p;
      gpureal p2 = p1;
      p1 = p;
      p = (x * (2 * n + 1) * p1 - (n + m) * p2) / (n - m + 1);
      YnmTheta = anm * ((n - m + 1) * p - (n + 1) * x * p1) / y;
      realj = ere * sourceShrd[6*i+0] - eim * sourceShrd[6*i+1];
      imagj = eim * sourceShrd[6*i+0] + ere * sourceShrd[6*i+1];
      spherical[0] += 2 * n / r * Ynm * realj;
      spherical[1] += 2 * YnmTheta * realj;
      spherical[2] -= 2 * m * Ynm * imagj;
      realj = ere * sourceShrd[6*i+2] - eim * sourceShrd[6*i+3];
      imagj = eim * sourceShrd[6*i+2] + ere * sourceShrd[6*i+3];
      spherical[3] += 2 * n / r * Ynm * realj;
      spherical[4] += 2 * YnmTheta * realj;
      spherical[5] -= 2 * m * Ynm * imagj;
      realj = ere * sourceShrd[6*i+4] - eim * sourceShrd[6*i+5];
      imagj = eim * sourceShrd[6*i+4] + ere * sourceShrd[6*i+5];
      spherical[6] += 2 * n / r * Ynm * realj;
      spherical[7] += 2 * YnmTheta * realj;
      spherical[8] -= 2 * m * Ynm * imagj;
      rhon *= r;
    }
    pn = -pn * fact * s;
    fact += 2;
  }
  sph2cart(r,theta,phi,&spherical[0],&cartesian[0]);
  sph2cart(r,theta,phi,&spherical[3],&cartesian[3]);
  sph2cart(r,theta,phi,&spherical[6],&cartesian[6]);
  target[0] += 0.25 / M_PI * (cartesian[5] - cartesian[7]);
  target[1] += 0.25 / M_PI * (cartesian[6] - cartesian[2]);
  target[2] += 0.25 / M_PI * (cartesian[1] - cartesian[3]);
}

__global__ void BiotSavartL2P_GPU(int *keysGlob, int *rangeGlob, gpureal *targetGlob, gpureal *sourceGlob) {
  int keys = keysGlob[blockIdx.x];
  int numList = rangeGlob[keys];
  gpureal targetX[3];
  gpureal target[3] = {0, 0, 0};
  __shared__ gpureal sourceShrd[6*THREADS];
  __shared__ gpureal factShrd[2*P];
  gpureal fact = 1;
  for( int i=0; i<2*P; ++i ) {
    factShrd[i] = fact;
    fact *= i + 1;
  }
  __syncthreads();
  int itarget = blockIdx.x * THREADS + threadIdx.x;
  targetX[0] = targetGlob[6*itarget+0];
  targetX[1] = targetGlob[6*itarget+1];
  targetX[2] = targetGlob[6*itarget+2];
  for( int ilist=0; ilist<numList; ++ilist ) {
    int begin = rangeGlob[keys+3*ilist+1];
    float3 d;
    d.x = targetX[0] - sourceGlob[begin+0];
    d.y = targetX[1] - sourceGlob[begin+1];
    d.z = targetX[2] - sourceGlob[begin+2];
    __syncthreads();
    if( threadIdx.x < NTERM ) {
      sourceShrd[6*threadIdx.x+0] = sourceGlob[begin+6*threadIdx.x+3];
      sourceShrd[6*threadIdx.x+1] = sourceGlob[begin+6*threadIdx.x+4];
      sourceShrd[6*threadIdx.x+2] = sourceGlob[begin+6*threadIdx.x+5];
      sourceShrd[6*threadIdx.x+3] = sourceGlob[begin+6*threadIdx.x+6];
      sourceShrd[6*threadIdx.x+4] = sourceGlob[begin+6*threadIdx.x+7];
      sourceShrd[6*threadIdx.x+5] = sourceGlob[begin+6*threadIdx.x+8];
    }
    __syncthreads();
    gpureal r,theta,phi;
    cart2sph(r,theta,phi,d.x,d.y,d.z);
    BiotSavartL2P_core(target,r,theta,phi,factShrd,sourceShrd);
  }
  targetGlob[6*itarget+0] = target[0];
  targetGlob[6*itarget+1] = target[1];
  targetGlob[6*itarget+2] = target[2];
}

template<>
void Kernel<BiotSavart>::L2P() {
  hipDeviceSynchronize();
  startTimer("L2P GPUkernel");
  int numBlocks = keysHost.size();\
  if( numBlocks != 0 ) {\
    BiotSavartL2P_GPU<<< numBlocks, THREADS >>>(keysDevc,rangeDevc,targetDevc,sourceDevc);\
  }\
  CUT_CHECK_ERROR("Kernel execution failed");\
  hipDeviceSynchronize();\
  stopTimer("L2P GPUkernel");\
}
