#include "hip/hip_runtime.h"
#include "kernel.h"
#include "gpu.h"

void Kernel::initialize() {
  hipSetDevice(MPIRANK % GPUS);                                // Set GPU device
  hipDeviceSynchronize();                                      // Sync GPU threads
}

void Kernel::P2M() {
  for( B_iter B=CJ->LEAF; B!=CJ->LEAF+CJ->NLEAF; ++B ) {
    vect dist = CJ->X - B->pos;
    CJ->M[0] += B->scal;
    CJ->M[1] += B->scal * dist[0];
    CJ->M[2] += B->scal * dist[1];
    CJ->M[3] += B->scal * dist[2];
    CJ->M[4] += B->scal * dist[0] * dist[0] / 2;
    CJ->M[5] += B->scal * dist[1] * dist[1] / 2;
    CJ->M[6] += B->scal * dist[2] * dist[2] / 2;
    CJ->M[7] += B->scal * dist[0] * dist[1];
    CJ->M[8] += B->scal * dist[1] * dist[2];
    CJ->M[9] += B->scal * dist[2] * dist[0];
  }
}

void Kernel::M2M() {
  vect dist = CI->X - CJ->X;
  CI->M[0] += CJ->M[0];
  CI->M[1] += CJ->M[1] +  dist[0] * CJ->M[0];
  CI->M[2] += CJ->M[2] +  dist[1] * CJ->M[0];
  CI->M[3] += CJ->M[3] +  dist[2] * CJ->M[0];
  CI->M[4] += CJ->M[4] +  dist[0] * CJ->M[1] + dist[0] * dist[0]  * CJ->M[0] / 2;
  CI->M[5] += CJ->M[5] +  dist[1] * CJ->M[2] + dist[1] * dist[1]  * CJ->M[0] / 2;
  CI->M[6] += CJ->M[6] +  dist[2] * CJ->M[3] + dist[2] * dist[2]  * CJ->M[0] / 2;
  CI->M[7] += CJ->M[7] + (dist[0] * CJ->M[2] + dist[1] * CJ->M[1] + dist[0] * dist[1] * CJ->M[0]) / 2;
  CI->M[8] += CJ->M[8] + (dist[1] * CJ->M[3] + dist[2] * CJ->M[2] + dist[1] * dist[2] * CJ->M[0]) / 2;
  CI->M[9] += CJ->M[9] + (dist[2] * CJ->M[1] + dist[0] * CJ->M[3] + dist[2] * dist[0] * CJ->M[0]) / 2;
}

void Kernel::M2L() {
  vect dist = CI->X - CJ->X;
  real R = std::sqrt(norm(dist));
  real R3 = R * R * R;
  real R5 = R3 * R * R;
  CI->L[0] += CJ->M[0] / R;
  CI->L[0] += CJ->M[1] * (-dist[0] / R3);
  CI->L[0] += CJ->M[2] * (-dist[1] / R3);
  CI->L[0] += CJ->M[3] * (-dist[2] / R3);
  CI->L[0] += CJ->M[4] * (3 * dist[0] * dist[0] / R5 - 1 / R3);
  CI->L[0] += CJ->M[5] * (3 * dist[1] * dist[1] / R5 - 1 / R3);
  CI->L[0] += CJ->M[6] * (3 * dist[2] * dist[2] / R5 - 1 / R3);
  CI->L[0] += CJ->M[7] * (3 * dist[0] * dist[1] / R5);
  CI->L[0] += CJ->M[8] * (3 * dist[1] * dist[2] / R5);
  CI->L[0] += CJ->M[9] * (3 * dist[2] * dist[0] / R5);
  CI->L[1] += CJ->M[0] * (-dist[0] / R3);
  CI->L[1] += CJ->M[1] * (3 * dist[0] * dist[0] / R5 - 1 / R3);
  CI->L[1] += CJ->M[2] * (3 * dist[0] * dist[1] / R5);
  CI->L[1] += CJ->M[3] * (3 * dist[0] * dist[2] / R5);
  CI->L[2] += CJ->M[0] * (-dist[1] / R3);
  CI->L[2] += CJ->M[1] * (3 * dist[1] * dist[0] / R5);
  CI->L[2] += CJ->M[2] * (3 * dist[1] * dist[1] / R5 - 1 / R3);
  CI->L[2] += CJ->M[3] * (3 * dist[1] * dist[2] / R5);
  CI->L[3] += CJ->M[0] * (-dist[2] / R3);
  CI->L[3] += CJ->M[1] * (3 * dist[2] * dist[0] / R5);
  CI->L[3] += CJ->M[2] * (3 * dist[2] * dist[1] / R5);
  CI->L[3] += CJ->M[3] * (3 * dist[2] * dist[2] / R5 - 1 / R3);
  CI->L[4] += CJ->M[0] * (3 * dist[0] * dist[0] / R5 - 1 / R3);
  CI->L[5] += CJ->M[0] * (3 * dist[1] * dist[1] / R5 - 1 / R3);
  CI->L[6] += CJ->M[0] * (3 * dist[2] * dist[2] / R5 - 1 / R3);
  CI->L[7] += CJ->M[0] * (3 * dist[0] * dist[1] / R5);
  CI->L[8] += CJ->M[0] * (3 * dist[1] * dist[2] / R5);
  CI->L[9] += CJ->M[0] * (3 * dist[2] * dist[0] / R5);
}

void Kernel::M2P() {
  for( B_iter B=CI->LEAF; B!=CI->LEAF+CI->NLEAF; ++B ) {
    vect dist = B->pos - CJ->X;
    real R = std::sqrt(norm(dist));
    real R3 = R * R * R;
    real R5 = R3 * R * R;
    B->pot += CJ->M[0] / R;
    B->pot += CJ->M[1] * (-dist[0] / R3);
    B->pot += CJ->M[2] * (-dist[1] / R3);
    B->pot += CJ->M[3] * (-dist[2] / R3);
    B->pot += CJ->M[4] * (3 * dist[0] * dist[0] / R5 - 1 / R3);
    B->pot += CJ->M[5] * (3 * dist[1] * dist[1] / R5 - 1 / R3);
    B->pot += CJ->M[6] * (3 * dist[2] * dist[2] / R5 - 1 / R3);
    B->pot += CJ->M[7] * (3 * dist[0] * dist[1] / R5);
    B->pot += CJ->M[8] * (3 * dist[1] * dist[2] / R5);
    B->pot += CJ->M[9] * (3 * dist[2] * dist[0] / R5);
  }
}

__global__ void P2P_GPU(float4 *sourceGlob, float *targetGlob) {
  int N = deviceConstant[0];
  float3 d;
  __shared__ float4 sourceShrd[THREADS];
  float4 target = sourceGlob[blockIdx.x * THREADS + threadIdx.x];
  target.w *= -rsqrtf(EPS2);
  for( int iblok=0; iblok<(N-1)/THREADS; iblok++) {
    __syncthreads();
    sourceShrd[threadIdx.x] = sourceGlob[iblok * THREADS + threadIdx.x];
    __syncthreads();
#pragma unroll 32
    for( int i=0; i<THREADS; i++ ) {
      d.x = target.x - sourceShrd[i].x;
      d.y = target.y - sourceShrd[i].y;
      d.z = target.z - sourceShrd[i].z;
      target.w += sourceShrd[i].w * rsqrtf(d.x * d.x + d.y * d.y + d.z * d.z + EPS2);
    }
  }
  int iblok = (N-1)/THREADS;
  __syncthreads();
  sourceShrd[threadIdx.x] = sourceGlob[iblok * THREADS + threadIdx.x];
  __syncthreads();
  for( int i=0; i<N - (iblok * THREADS); i++ ) {
    d.x = target.x - sourceShrd[i].x;
    d.y = target.y - sourceShrd[i].y;
    d.z = target.z - sourceShrd[i].z;
    target.w += sourceShrd[i].w * rsqrtf(d.x * d.x + d.y * d.y + d.z * d.z + EPS2);
  }
  targetGlob[blockIdx.x * THREADS + threadIdx.x] = target.w;
}

void Kernel::P2P(float4 *sourceDevc, float *targetDevc) {
  P2P_GPU<<< Nround/THREADS, THREADS >>>(sourceDevc,targetDevc);
}

void Kernel::L2L() {
  vect dist = CI->X - CJ->X;
  for( int i=0; i<10; ++i )
    CI->L[i] += CJ->L[i];
  CI->L[0] += CJ->L[1] * dist[0];
  CI->L[0] += CJ->L[2] * dist[1];
  CI->L[0] += CJ->L[3] * dist[2];
  CI->L[0] += CJ->L[4] * dist[0] * dist[0] / 2;
  CI->L[0] += CJ->L[5] * dist[1] * dist[1] / 2;
  CI->L[0] += CJ->L[6] * dist[2] * dist[2] / 2;
  CI->L[0] += CJ->L[7] * dist[0] * dist[1];
  CI->L[0] += CJ->L[8] * dist[1] * dist[2];
  CI->L[0] += CJ->L[9] * dist[2] * dist[0];
  CI->L[1] += CJ->L[4] * dist[0] * dist[0] / 2;
  CI->L[1] += CJ->L[7] * dist[0] * dist[1];
  CI->L[1] += CJ->L[9] * dist[0] * dist[2];
  CI->L[2] += CJ->L[7] * dist[1] * dist[0];
  CI->L[2] += CJ->L[5] * dist[1] * dist[1] / 2;
  CI->L[2] += CJ->L[8] * dist[1] * dist[2];
  CI->L[3] += CJ->L[9] * dist[2] * dist[0];
  CI->L[3] += CJ->L[8] * dist[2] * dist[1];
  CI->L[3] += CJ->L[6] * dist[2] * dist[2] / 2;
}

void Kernel::L2P() {
  for( B_iter B=CI->LEAF; B!=CI->LEAF+CI->NLEAF; ++B ) {
    vect dist = B->pos - CI->X;
    B->pot += CI->L[0];
    B->pot += CI->L[1] * dist[0];
    B->pot += CI->L[2] * dist[1];
    B->pot += CI->L[3] * dist[2];
    B->pot += CI->L[4] * dist[0] * dist[0] / 2;
    B->pot += CI->L[5] * dist[1] * dist[1] / 2;
    B->pot += CI->L[6] * dist[2] * dist[2] / 2;
    B->pot += CI->L[7] * dist[0] * dist[1];
    B->pot += CI->L[8] * dist[1] * dist[2];
    B->pot += CI->L[9] * dist[2] * dist[0];
  }
}

void Kernel::finalize() {}
