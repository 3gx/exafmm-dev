#include "hip/hip_runtime.h"
/*
Copyright (C) 2011 by Rio Yokota, Simon Layton, Lorena Barba

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
THE SOFTWARE.
*/
#define KERNEL
#include "kernel.hpp"
#undef KERNEL
#  define CUDA_SAFE_CALL( call) {                                    \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    } }

#ifdef _DEBUG
#  define CUT_CHECK_ERROR(errorMessage) {                                    \
    hipError_t err = hipGetLastError();                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",    \
                errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );\
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
    err = CUT_DEVICE_SYNCHRONIZE();                                           \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",    \
                errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );\
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
    }
#else
#  define CUT_CHECK_ERROR(errorMessage) {                                    \
    hipError_t err = hipGetLastError();                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",    \
                errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );\
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
    }
#endif

__device__ __constant__ gpureal constDevc[514];                 // Constants on device

namespace                                                       // Prevent overlap of definitions among equations
{
__device__ void cart2sph(gpureal& r, gpureal& theta, gpureal& phi,// Get r,theta,phi from x,y,z on GPU
                         gpureal dx, gpureal dy, gpureal dz)
{
    r = sqrtf(dx * dx + dy * dy + dz * dz) + EPS;                 // r = sqrt(x^2 + y^2 + z^2) + eps
    theta = acosf(dz / r);                                        // theta = acos(z / r)
    if ( fabs(dx) + fabs(dy) < EPS )                              // If |x| < eps & |y| < eps
    {
        phi = 0;                                                    //  phi can be anything so we set it to 0
    }
    else if ( fabs(dx) < EPS )                                  // If |x| < eps
    {
        phi = dy / fabs(dy) * M_PI * 0.5;                           //  phi = sign(y) * pi / 2
    }
    else if ( dx > 0 )                                          // If x > 0
    {
        phi = atanf(dy / dx);                                       //  phi = atan(y / x)
    }
    else                                                        // If x < 0
    {
        phi = atanf(dy / dx) + M_PI;                                //  phi = atan(y / x) + pi
    }                                                             // End if for x,y cases
}

__device__ void sph2cart(gpureal r, gpureal theta, gpureal phi, // Spherical to cartesian coordinates on GPU
                         gpureal *spherical, gpureal *cartesian)
{
    cartesian[0] = sinf(theta) * cosf(phi) * spherical[0]         // x component (not x itself)
                   + cosf(theta) * cosf(phi) / r * spherical[1]
                   - sinf(phi) / r / sinf(theta) * spherical[2];
    cartesian[1] = sinf(theta) * sinf(phi) * spherical[0]         // y component (not y itself)
                   + cosf(theta) * sinf(phi) / r * spherical[1]
                   + cosf(phi) / r / sinf(theta) * spherical[2];
    cartesian[2] = cosf(theta) * spherical[0]                     // z component (not z itself)
                   - sinf(theta) / r * spherical[1];
}

__device__ void evalMultipole(gpureal *YnmShrd, gpureal rho,    // Evaluate solid harmonics r^n * Ynm on GPU
                              gpureal alpha, gpureal *factShrd)
{
    gpureal x = cosf(alpha);                                      // x = cos(alpha)
    gpureal y = sinf(alpha);                                      // y = sin(alpha)
    gpureal fact = 1;                                             // Initialize 2 * m + 1
    gpureal pn = 1;                                               // Initialize Legendre polynomial Pn
    gpureal rhom = 1;                                             // Initialize rho^m
    for ( int m = 0; m < P; ++m )                                 // Loop over m in Ynm
    {
        gpureal p = pn;                                             //  Associate Legendre polynomial Pnm
        int npn = m * m + 2 * m;                                    //  Index of Ynm for m > 0
        int nmn = m * m;                                            //  Index of Ynm for m < 0
        YnmShrd[npn] = rhom * p / factShrd[2*m];                    //  rho^m * Ynm for m > 0
        YnmShrd[nmn] = YnmShrd[npn];                                //  Use conjugate relation for m < 0
        gpureal p1 = p;                                             //  Pnm-1
        p = x * (2 * m + 1) * p;                                    //  Pnm using recurrence relation
        rhom *= -rho;                                               //  rho^m
        gpureal rhon = rhom;                                        //  rho^n
        for ( int n = m + 1; n < P; ++n )                           //  Loop over n in Yn
        {
            int npm = n * n + n + m;                                  //   Index of Ynm for m > 0
            int nmm = n * n + n - m;                                  //   Index of Ynm for m < 0
            YnmShrd[npm] = rhon * p / factShrd[n+m];                  //   rho^n * Ynm
            YnmShrd[nmm] = YnmShrd[npm];                              //   Use conjugate relation for m < 0
            gpureal p2 = p1;                                          //   Pnm-2
            p1 = p;                                                   //   Pnm-1
            p = (x * (2 * n + 1) * p1 - (n + m) * p2) / (n - m + 1);  //   Pnm using recurrence relation
            rhon *= -rho;                                             //   Update rho^n
        }                                                           //  End loop over n in Ynm
        pn = -pn * fact * y;                                        //  Pn
        fact += 2;                                                  //  2 * m + 1
    }                                                             // End loop over m in Ynm
}

__device__ void evalLocal(gpureal *YnmShrd, gpureal rho,        // Evaluate singular harmonics r^(-n-1) * Ynm
                          gpureal alpha, gpureal *factShrd)
{
    gpureal x = cosf(alpha);                                      // x = cos(alpha)
    gpureal y = sinf(alpha);                                      // y = sin(alpha)
    gpureal rho_1 = 1 / rho;                                      // 1 / rho
    for ( int l = threadIdx.x; l < (2*P + 1)*P; l += THREADS )    // Loop over coefficients in Ynm
    {
        gpureal fact = 1;                                           //  Initialize 2 * m + 1
        gpureal pn = 1;                                             //  Initialize Legendre polynomial Pn
        gpureal rhom = rho_1;                                       //  Initialize rho^(-m-1)
        int nn = floor(sqrtf(2 * l + 0.25) - 0.5);                  //  Calculate index n of Ynm
        int mm = 0;                                                 //  Initialize index m of Ynm
        gpureal Ynm;                                                //  Define temporary Ynm
        for ( int i = 0; i <= nn; ++i ) mm += i;                    //  Offset of m
        mm = l - mm;                                                //  Calculate index m of Ynm
        int n;                                                      //  Define temporary n
        for ( int m = 0; m < mm; ++m )                              //  Loop up to m
        {
            rhom *= rho_1;                                            //   rho^(-m-1)
            pn = -pn * fact * y;                                      //   Pn
            fact += 2;                                                //   2 * m + 1
        }                                                           //  End loop up to m
        int m = mm;                                                 //  Define temporary m
        gpureal p = pn;                                             //  Associated Legendre polynomial Pnm
        if ( mm == nn ) Ynm = rhom * p * EPS;                       //  Ynm for n == m
        gpureal p1 = p;                                             //  Pnm-1
        p = x * (2 * m + 1) * p;                                    //  Pnm
        rhom *= rho_1;                                              //  rho^(-m-1)
        gpureal rhon = rhom;                                        //  rho^(-n-1)
        for ( n = m + 1; n < nn; ++n )                              //  Loop up to n
        {
            gpureal p2 = p1;                                          //   Pnm-2
            p1 = p;                                                   //   Pnm-1
            p = (x * (2 * n + 1) * p1 - (n + m) * p2) / (n - m + 1);  //   Pnm
            rhon *= rho_1;                                            //   rho^(-n-1)
        }                                                           //  End loop up to n
        if ( n <= nn ) Ynm = rhon * p * factShrd[n-m];              //  rho^(-n-1) * Ynm
        YnmShrd[l] = Ynm;                                           //  Put Ynm in shared memory
    }                                                             // End loop over coefficients in Ynm
    __syncthreads();                                              // Syncronize threads
}
}                                                               // End anonymous namespace

void Kernel<Stokes>::initialize()
{
    startTimer("Init GPU     ");                                  // Start timer
    hipDeviceReset();                                             // Exit GPU thread
    hipSetDevice(DEVICE);                                        // Set GPU device
    hipDeviceSynchronize();                                      // Sync GPU threads
#ifdef CUDA_4_1
    hipSetDeviceFlags(hipDeviceMapHost);
#endif
    stopTimer("Init GPU     ", MPIRANK == 0);                     // Stop timer & print
    eraseTimer("Init GPU     ");                                  // Erase timer
}

void Kernel<Stokes>::finalize()
{
}

void Kernel<Stokes>::allocate()
{
    hipDeviceSynchronize();
    startTimer("hipMalloc   ");
#ifdef CUDA_4_1
    if ( keysHost.size() > keysDevcSize )
    {
        CUDA_SAFE_CALL(hipHostRegister(&keysHost[0], sizeof(int) * keysHost.size(), hipHostRegisterMapped));
        CUDA_SAFE_CALL(hipHostGetDevicePointer((void **) &keysDevc, (void *)&keysHost[0], 0));
        keysDevcSize = keysHost.size();
    }
    if ( rangeHost.size() > rangeDevcSize )
    {
        CUDA_SAFE_CALL(hipHostRegister(&rangeHost[0], sizeof(int) * rangeHost.size(), hipHostRegisterMapped));
        CUDA_SAFE_CALL(hipHostGetDevicePointer((void **) &rangeDevc, (void *)&rangeHost[0], 0));
        rangeDevcSize = rangeHost.size();
    }
    if ( sourceHost.size() > sourceDevcSize )
    {
        CUDA_SAFE_CALL(hipHostRegister(&sourceDevc[0], sizeof(gpureal) * sourceHost.size(), hipHostRegisterMapped));
        CUDA_SAFE_CALL(hipHostGetDevicePointer((void **) &sourceDevc, (void *)&sourceDevc[0], 0));
        sourceDevcSize = sourceHost.size();
    }
    if ( targetHost.size() > targetDevcSize )
    {
        CUDA_SAFE_CALL(hipHostRegister(&targetHost[0], sizeof(gpureal) * targetHost.size(), hipHostRegisterMapped));
        CUDA_SAFE_CALL(hipHostGetDevicePointer((void **) &targetDevc, (void *)&targetHost[0], 0));
        targetDevcSize = targetHost.size();
    }
#else
    if ( keysHost.size() > keysDevcSize )
    {
        if ( keysDevcSize != 0 ) CUDA_SAFE_CALL(hipFree(keysDevc));
        CUDA_SAFE_CALL(hipMalloc( (void**) &keysDevc, keysHost.size()*sizeof(int) ));
        keysDevcSize = keysHost.size();
    }
    if ( rangeHost.size() > rangeDevcSize )
    {
        if ( rangeDevcSize != 0 ) CUDA_SAFE_CALL(hipFree(rangeDevc));
        CUDA_SAFE_CALL(hipMalloc( (void**) &rangeDevc, rangeHost.size()*sizeof(int) ));
        rangeDevcSize = rangeHost.size();
    }
    if ( sourceHost.size() > sourceDevcSize )
    {
        if ( sourceDevcSize != 0 ) CUDA_SAFE_CALL(hipFree(sourceDevc));
        CUDA_SAFE_CALL(hipMalloc( (void**) &sourceDevc, sourceHost.size()*sizeof(gpureal) ));
        sourceDevcSize = sourceHost.size();
    }
    if ( targetHost.size() > targetDevcSize )
    {
        if ( targetDevcSize != 0 ) CUDA_SAFE_CALL(hipFree(targetDevc));
        CUDA_SAFE_CALL(hipMalloc( (void**) &targetDevc, targetHost.size()*sizeof(gpureal) ));
        targetDevcSize = targetHost.size();
    }
#endif
    hipDeviceSynchronize();
    stopTimer("hipMalloc   ");
}


void Kernel<Stokes>::hostToDevice()
{
    hipDeviceSynchronize();
    startTimer("hipMemcpy   ");
#ifndef CUDA_4_1
    CUDA_SAFE_CALL(hipMemcpy(keysDevc,  &keysHost[0],  keysHost.size()*sizeof(int), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(rangeDevc, &rangeHost[0], rangeHost.size()*sizeof(int), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(sourceDevc, &sourceHost[0], sourceHost.size()*sizeof(gpureal), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(targetDevc, &targetHost[0], targetHost.size()*sizeof(gpureal), hipMemcpyHostToDevice));
#endif
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constDevc), &constHost[0], constHost.size()*sizeof(gpureal)));
    hipDeviceSynchronize();
    stopTimer("hipMemcpy   ");
}

void Kernel<Stokes>::deviceToHost()
{
    hipDeviceSynchronize();
    startTimer("hipMemcpy   ");
#ifdef CUDA_4_1
    CUDA_SAFE_CALL(hipHostUnregister(&keysHost[0]));
    CUDA_SAFE_CALL(hipHostUnregister(&rangeHost[0]));
    CUDA_SAFE_CALL(hipHostUnregister(&sourceHost[0]));
    CUDA_SAFE_CALL(hipHostUnregister(&targetHost[0]));
#else
    CUDA_SAFE_CALL(hipMemcpy(&targetHost[0], targetDevc, targetHost.size()*sizeof(gpureal), hipMemcpyDeviceToHost));
#endif
    hipDeviceSynchronize();
    stopTimer("hipMemcpy   ");
}

__device__ void StokesP2P_core(gpureal *target, gpureal *targetX, gpureal *sourceShrd, float3 d, int i, float delta)
{
    d.x += targetX[0];
    d.x -= sourceShrd[6*i+0];
    d.y += targetX[1];
    d.y -= sourceShrd[6*i+1];
    d.z += targetX[2];
    d.z -= sourceShrd[6*i+2];

    float3 force = {sourceShrd[6*i+3], sourceShrd[6*i+4], sourceShrd[6*i+5]};

    float r2 = d.x * d.x + d.y * d.y + d.z * d.z;
    float d2 = delta * delta;
    float R1 = r2 + d2;
    float R2 = R1 + d2;
    float invR = 1.0 / R1;
    float H = sqrt(invR) * invR;

    float fdx =  force.x * d.x + force.y * d.y + force.z * d.z;

    target[0] += H * (force.x * R2 + fdx * d.x);
    target[1] += H * (force.y * R2 + fdx * d.y);
    target[2] += H * (force.z * R2 + fdx * d.z);

}

__global__ void StokesP2P_GPU(int *keysGlob, int *rangeGlob, gpureal *targetGlob, gpureal *sourceGlob, float delta)
{
    int keys = keysGlob[blockIdx.x];
    int numList = rangeGlob[keys];
    gpureal D0 = -constDevc[0];
    gpureal targetX[3];
    gpureal target[4] = {0, 0, 0, 0};
    __shared__ gpureal sourceShrd[6*THREADS];
    int itarget = blockIdx.x * THREADS + threadIdx.x;
    targetX[0] = targetGlob[4*itarget+0];
    targetX[1] = targetGlob[4*itarget+1];
    targetX[2] = targetGlob[4*itarget+2];
    for ( int ilist = 0; ilist < numList; ++ilist )
    {
        int begin     = rangeGlob[keys+3*ilist+1];
        int size      = rangeGlob[keys+3*ilist+2];
        int Iperiodic = rangeGlob[keys+3*ilist+3];
        for ( int iblok = 0; iblok < (size - 1) / THREADS; ++iblok ){
            int isource = begin + iblok * THREADS + threadIdx.x;
            __syncthreads();
            sourceShrd[6*threadIdx.x+0] = sourceGlob[6*isource+0];
            sourceShrd[6*threadIdx.x+1] = sourceGlob[6*isource+1];
            sourceShrd[6*threadIdx.x+2] = sourceGlob[6*isource+2];
            sourceShrd[6*threadIdx.x+3] = sourceGlob[6*isource+3];
            sourceShrd[6*threadIdx.x+4] = sourceGlob[6*isource+4];
            sourceShrd[6*threadIdx.x+5] = sourceGlob[6*isource+5];
            __syncthreads();
            int I = 0;
            for ( int ix = -1; ix <= 1; ++ix ){
                for ( int iy = -1; iy <= 1; ++iy ){
                    for ( int iz = -1; iz <= 1; ++iz, ++I ){
                        if ( Iperiodic & (1 << I) )
                        {
                            float3 d;
                            d.x = ix * D0;
                            d.y = iy * D0;
                            d.z = iz * D0;
#pragma unroll 64
                            for ( int i = 0; i < THREADS; ++i ){
                                StokesP2P_core(target, targetX, sourceShrd, d, i, delta);
                            }
                        }
                    }
                }
            }
        }
        int iblok = (size - 1) / THREADS;
        int isource = begin + iblok * THREADS + threadIdx.x;
        __syncthreads();
        if ( threadIdx.x < size - iblok * THREADS )
        {
            sourceShrd[6*threadIdx.x+0] = sourceGlob[6*isource+0];
            sourceShrd[6*threadIdx.x+1] = sourceGlob[6*isource+1];
            sourceShrd[6*threadIdx.x+2] = sourceGlob[6*isource+2];
            sourceShrd[6*threadIdx.x+3] = sourceGlob[6*isource+3];
            sourceShrd[6*threadIdx.x+4] = sourceGlob[6*isource+4];
            sourceShrd[6*threadIdx.x+5] = sourceGlob[6*isource+5];
        }
        __syncthreads();
        int I = 0;
        int icounter = 0;
        for ( int ix = -1; ix <= 1; ++ix ){
            for ( int iy = -1; iy <= 1; ++iy ){
                for ( int iz = -1; iz <= 1; ++iz, ++I ){
                    if ( Iperiodic & (1 << I) )
                    {
                        icounter++;
                        float3 d;
                        d.x = ix * D0;
                        d.y = iy * D0;
                        d.z = iz * D0;
                        for ( int i = 0; i < size - iblok*THREADS; ++i )
                        {
                            StokesP2P_core(target, targetX, sourceShrd, d, i, delta);
                        }
                    }
                }
            }
        }
    }
    targetGlob[4*itarget+0] = target[0];
    targetGlob[4*itarget+1] = target[1];
    targetGlob[4*itarget+2] = target[2];
    targetGlob[4*itarget+3] = target[3];
}

void Kernel<Stokes>::P2P()
{
    hipDeviceSynchronize();
    startTimer("P2P GPUkernel");
    int numBlocks = keysHost.size();
    if ( numBlocks != 0 )
    {
        StokesP2P_GPU <<< numBlocks, THREADS >>>(keysDevc, rangeDevc, targetDevc, sourceDevc, delta);
    }
    CUT_CHECK_ERROR("Kernel execution failed");
    hipDeviceSynchronize();
    stopTimer("P2P GPUkernel");
}


