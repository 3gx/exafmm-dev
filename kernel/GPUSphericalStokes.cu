#include "hip/hip_runtime.h"
/*
Copyright (C) 2011 by Rio Yokota, Simon Layton, Lorena Barba

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
THE SOFTWARE.
*/
#define KERNEL
#include "kernel.h"
#undef KERNEL
#  define CUDA_SAFE_CALL( call) {                                    \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    } }

#ifdef _DEBUG
#  define CUT_CHECK_ERROR(errorMessage) {                                    \
    hipError_t err = hipGetLastError();                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",    \
                errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );\
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
    err = CUT_DEVICE_SYNCHRONIZE();                                           \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",    \
                errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );\
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
    }
#else
#  define CUT_CHECK_ERROR(errorMessage) {                                    \
    hipError_t err = hipGetLastError();                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",    \
                errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );\
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
    }
#endif

__device__ __constant__ gpureal constDevc[514];                 // Constants on device

void Kernel<Stokes>::initialize()
{
    startTimer("Init GPU     ");                                  // Start timer
    hipDeviceReset();                                             // Exit GPU thread
    hipSetDevice(DEVICE);                                        // Set GPU device
    hipDeviceSynchronize();                                      // Sync GPU threads
#ifdef CUDA_4_1
    hipSetDeviceFlags(hipDeviceMapHost);
#endif
    stopTimer("Init GPU     ", MPIRANK == 0);                     // Stop timer & print
    eraseTimer("Init GPU     ");                                  // Erase timer
}

void Kernel<Stokes>::finalize()
{
}

void Kernel<Stokes>::allocate()
{
    hipDeviceSynchronize();
    startTimer("hipMalloc   ");
#ifdef CUDA_4_1
    if ( keysHost.size() > keysDevcSize )
    {
        CUDA_SAFE_CALL(hipHostRegister(&keysHost[0], sizeof(int) * keysHost.size(), hipHostRegisterMapped));
        CUDA_SAFE_CALL(hipHostGetDevicePointer((void **) &keysDevc, (void *)&keysHost[0], 0));
        keysDevcSize = keysHost.size();
    }
    if ( rangeHost.size() > rangeDevcSize )
    {
        CUDA_SAFE_CALL(hipHostRegister(&rangeHost[0], sizeof(int) * rangeHost.size(), hipHostRegisterMapped));
        CUDA_SAFE_CALL(hipHostGetDevicePointer((void **) &rangeDevc, (void *)&rangeHost[0], 0));
        rangeDevcSize = rangeHost.size();
    }
    if ( sourceHost.size() > sourceDevcSize )
    {
        CUDA_SAFE_CALL(hipHostRegister(&sourceDevc[0], sizeof(gpureal) * sourceHost.size(), hipHostRegisterMapped));
        CUDA_SAFE_CALL(hipHostGetDevicePointer((void **) &sourceDevc, (void *)&sourceDevc[0], 0));
        sourceDevcSize = sourceHost.size();
    }
    if ( targetHost.size() > targetDevcSize )
    {
        CUDA_SAFE_CALL(hipHostRegister(&targetHost[0], sizeof(gpureal) * targetHost.size(), hipHostRegisterMapped));
        CUDA_SAFE_CALL(hipHostGetDevicePointer((void **) &targetDevc, (void *)&targetHost[0], 0));
        targetDevcSize = targetHost.size();
    }
#else
    if ( keysHost.size() > keysDevcSize )
    {
        if ( keysDevcSize != 0 ) CUDA_SAFE_CALL(hipFree(keysDevc));
        CUDA_SAFE_CALL(hipMalloc( (void**) &keysDevc, keysHost.size()*sizeof(int) ));
        keysDevcSize = keysHost.size();
    }
    if ( rangeHost.size() > rangeDevcSize )
    {
        if ( rangeDevcSize != 0 ) CUDA_SAFE_CALL(hipFree(rangeDevc));
        CUDA_SAFE_CALL(hipMalloc( (void**) &rangeDevc, rangeHost.size()*sizeof(int) ));
        rangeDevcSize = rangeHost.size();
    }
    if ( sourceHost.size() > sourceDevcSize )
    {
        if ( sourceDevcSize != 0 ) CUDA_SAFE_CALL(hipFree(sourceDevc));
        CUDA_SAFE_CALL(hipMalloc( (void**) &sourceDevc, sourceHost.size()*sizeof(gpureal) ));
        sourceDevcSize = sourceHost.size();
    }
    if ( targetHost.size() > targetDevcSize )
    {
        if ( targetDevcSize != 0 ) CUDA_SAFE_CALL(hipFree(targetDevc));
        CUDA_SAFE_CALL(hipMalloc( (void**) &targetDevc, targetHost.size()*sizeof(gpureal) ));
        targetDevcSize = targetHost.size();
    }
#endif
    hipDeviceSynchronize();
    stopTimer("hipMalloc   ");
}


void Kernel<Stokes>::hostToDevice()
{
    hipDeviceSynchronize();
    startTimer("hipMemcpy   ");
#ifndef CUDA_4_1
    CUDA_SAFE_CALL(hipMemcpy(keysDevc,  &keysHost[0],  keysHost.size()*sizeof(int), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(rangeDevc, &rangeHost[0], rangeHost.size()*sizeof(int), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(sourceDevc, &sourceHost[0], sourceHost.size()*sizeof(gpureal), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(targetDevc, &targetHost[0], targetHost.size()*sizeof(gpureal), hipMemcpyHostToDevice));
#endif
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constDevc), &constHost[0], constHost.size()*sizeof(gpureal)));
    hipDeviceSynchronize();
    stopTimer("hipMemcpy   ");
}

void Kernel<Stokes>::deviceToHost()
{
    hipDeviceSynchronize();
    startTimer("hipMemcpy   ");
#ifdef CUDA_4_1
    CUDA_SAFE_CALL(hipHostUnregister(&keysHost[0]));
    CUDA_SAFE_CALL(hipHostUnregister(&rangeHost[0]));
    CUDA_SAFE_CALL(hipHostUnregister(&sourceHost[0]));
    CUDA_SAFE_CALL(hipHostUnregister(&targetHost[0]));
#else
    CUDA_SAFE_CALL(hipMemcpy(&targetHost[0], targetDevc, targetHost.size()*sizeof(gpureal), hipMemcpyDeviceToHost));
#endif
    hipDeviceSynchronize();
    stopTimer("hipMemcpy   ");
}

__device__ void StokesP2P_core(gpureal *target, gpureal *targetX, gpureal *sourceShrd, float3 d, int i, float delta)
{
    d.x += targetX[0];
    d.x -= sourceShrd[6*i+0];
    d.y += targetX[1];
    d.y -= sourceShrd[6*i+1];
    d.z += targetX[2];
    d.z -= sourceShrd[6*i+2];

    float3 force = {sourceShrd[6*i+3], sourceShrd[6*i+4], sourceShrd[6*i+5]};

    float r2 = d.x * d.x + d.y * d.y + d.z * d.z;
    float d2 = delta * delta;
    float R1 = r2 + d2;
    float R2 = R1 + d2;
    float invR = 1.0 / R1;
    float H = sqrt(invR) * invR;

    float fdx =  force.x * d.x + force.y * d.y + force.z * d.z;

    target[0] += H * (force.x * R2 + fdx * d.x);
    target[1] += H * (force.y * R2 + fdx * d.y);
    target[2] += H * (force.z * R2 + fdx * d.z);

}

__global__ void StokesP2P_GPU(int *keysGlob, int *rangeGlob, gpureal *targetGlob, gpureal *sourceGlob, float delta)
{
    int keys = keysGlob[blockIdx.x];
    int numList = rangeGlob[keys];
    gpureal D0 = -constDevc[0];
    gpureal targetX[3];
    gpureal target[4] = {0, 0, 0, 0};
    __shared__ gpureal sourceShrd[6*THREADS];
    int itarget = blockIdx.x * THREADS + threadIdx.x;
    targetX[0] = targetGlob[4*itarget+0];
    targetX[1] = targetGlob[4*itarget+1];
    targetX[2] = targetGlob[4*itarget+2];
    for ( int ilist = 0; ilist < numList; ++ilist )
    {
        int begin     = rangeGlob[keys+3*ilist+1];
        int size      = rangeGlob[keys+3*ilist+2];
        int Iperiodic = rangeGlob[keys+3*ilist+3];
        for ( int iblok = 0; iblok < (size - 1) / THREADS; ++iblok ){
            int isource = begin + iblok * THREADS + threadIdx.x;
            __syncthreads();
            sourceShrd[6*threadIdx.x+0] = sourceGlob[6*isource+0];
            sourceShrd[6*threadIdx.x+1] = sourceGlob[6*isource+1];
            sourceShrd[6*threadIdx.x+2] = sourceGlob[6*isource+2];
            sourceShrd[6*threadIdx.x+3] = sourceGlob[6*isource+3];
            sourceShrd[6*threadIdx.x+4] = sourceGlob[6*isource+4];
            sourceShrd[6*threadIdx.x+5] = sourceGlob[6*isource+5];
            __syncthreads();
            int I = 0;
            for ( int ix = -1; ix <= 1; ++ix ){
                for ( int iy = -1; iy <= 1; ++iy ){
                    for ( int iz = -1; iz <= 1; ++iz, ++I ){
                        if ( Iperiodic & (1 << I) )
                        {
                            float3 d;
                            d.x = ix * D0;
                            d.y = iy * D0;
                            d.z = iz * D0;
#pragma unroll 64
                            for ( int i = 0; i < THREADS; ++i ){
                                StokesP2P_core(target, targetX, sourceShrd, d, i, delta);
                            }
                        }
                    }
                }
            }
        }
        int iblok = (size - 1) / THREADS;
        int isource = begin + iblok * THREADS + threadIdx.x;
        __syncthreads();
        if ( threadIdx.x < size - iblok * THREADS )
        {
            sourceShrd[6*threadIdx.x+0] = sourceGlob[6*isource+0];
            sourceShrd[6*threadIdx.x+1] = sourceGlob[6*isource+1];
            sourceShrd[6*threadIdx.x+2] = sourceGlob[6*isource+2];
            sourceShrd[6*threadIdx.x+3] = sourceGlob[6*isource+3];
            sourceShrd[6*threadIdx.x+4] = sourceGlob[6*isource+4];
            sourceShrd[6*threadIdx.x+5] = sourceGlob[6*isource+5];
        }
        __syncthreads();
        int I = 0;
        int icounter = 0;
        for ( int ix = -1; ix <= 1; ++ix ){
            for ( int iy = -1; iy <= 1; ++iy ){
                for ( int iz = -1; iz <= 1; ++iz, ++I ){
                    if ( Iperiodic & (1 << I) )
                    {
                        icounter++;
                        float3 d;
                        d.x = ix * D0;
                        d.y = iy * D0;
                        d.z = iz * D0;
                        for ( int i = 0; i < size - iblok*THREADS; ++i )
                        {
                            StokesP2P_core(target, targetX, sourceShrd, d, i, delta);
                        }
                    }
                }
            }
        }
    }
    targetGlob[4*itarget+0] = target[0];
    targetGlob[4*itarget+1] = target[1];
    targetGlob[4*itarget+2] = target[2];
    targetGlob[4*itarget+3] = target[3];
}

void Kernel<Stokes>::P2P()
{
    hipDeviceSynchronize();
    startTimer("P2P GPUkernel");
    int numBlocks = keysHost.size();
    if ( numBlocks != 0 )
    {
        StokesP2P_GPU <<< numBlocks, THREADS >>>(keysDevc, rangeDevc, targetDevc, sourceDevc, delta);
    }
    CUT_CHECK_ERROR("Kernel execution failed");
    hipDeviceSynchronize();
    stopTimer("P2P GPUkernel");
}


