#include "hip/hip_runtime.h"
#include "octree.h"

static __device__ void pairMinMax(float3 &xmin, float3 &xmax,
                                  float4 reg_min, float4 reg_max) {
  xmin.x = fminf(xmin.x, reg_min.x);
  xmin.y = fminf(xmin.y, reg_min.y);
  xmin.z = fminf(xmin.z, reg_min.z);
  xmax.x = fmaxf(xmax.x, reg_max.x);
  xmax.y = fmaxf(xmax.y, reg_max.y);
  xmax.z = fmaxf(xmax.z, reg_max.z);
}

static __device__ void pairMinMax(int i, int j, float3 &xmin, float3 &xmax,
                                  volatile float3 *sh_xmin, volatile float3 *sh_xmax) {
  sh_xmin[i].x = xmin.x = fminf(xmin.x, sh_xmin[j].x);
  sh_xmin[i].y = xmin.y = fminf(xmin.y, sh_xmin[j].y);
  sh_xmin[i].z = xmin.z = fminf(xmin.z, sh_xmin[j].z);
  sh_xmax[i].x = xmax.x = fmaxf(xmax.x, sh_xmax[j].x);
  sh_xmax[i].y = xmax.y = fmaxf(xmax.y, sh_xmax[j].y);
  sh_xmax[i].z = xmax.z = fmaxf(xmax.z, sh_xmax[j].z);
}

static __device__ void sharedMinMax(float3 &xmin, float3 &xmax) {
  volatile __shared__ float3 sh_xmin[NCRIT];
  volatile __shared__ float3 sh_xmax[NCRIT];
  sh_xmin[threadIdx.x].x = xmin.x;
  sh_xmin[threadIdx.x].y = xmin.y;
  sh_xmin[threadIdx.x].z = xmin.z;
  sh_xmax[threadIdx.x].x = xmax.x;
  sh_xmax[threadIdx.x].y = xmax.y;
  sh_xmax[threadIdx.x].z = xmax.z;

  __syncthreads();
  if(blockDim.x >= 512 && threadIdx.x < 256)
    pairMinMax(threadIdx.x, threadIdx.x + 256, xmin, xmax, sh_xmin, sh_xmax);
  __syncthreads();
  if(blockDim.x >= 256 && threadIdx.x < 128)
    pairMinMax(threadIdx.x, threadIdx.x + 128, xmin, xmax, sh_xmin, sh_xmax);
  __syncthreads();
  if(blockDim.x >= 128 && threadIdx.x < 64)
    pairMinMax(threadIdx.x, threadIdx.x + 64, xmin, xmax, sh_xmin, sh_xmax);
  __syncthreads();
  if(blockDim.x >= 64 && threadIdx.x < 32)
    pairMinMax(threadIdx.x, threadIdx.x + 32, xmin, xmax, sh_xmin, sh_xmax);
  if(blockDim.x >= 32 && threadIdx.x < 16)
    pairMinMax(threadIdx.x, threadIdx.x + 16, xmin, xmax, sh_xmin, sh_xmax);
  if(threadIdx.x < 8) {
    pairMinMax(threadIdx.x, threadIdx.x +  8, xmin, xmax, sh_xmin, sh_xmax);
    pairMinMax(threadIdx.x, threadIdx.x +  4, xmin, xmax, sh_xmin, sh_xmax);
    pairMinMax(threadIdx.x, threadIdx.x +  2, xmin, xmax, sh_xmin, sh_xmax);
    pairMinMax(threadIdx.x, threadIdx.x +  1, xmin, xmax, sh_xmin, sh_xmax);
  }
}

static __device__ uint4 getKey(int4 index3) {
  const int bits = 30;
  const int C[8] = {0, 1, 7, 6, 3, 2, 4, 5};
  uint4 key4 = {0, 0, 0, 0};
  int mask = 1 << (bits - 1);
  int key = 0;
  for( int i=0; i<bits; i++, mask >>= 1) {
    int xi = (index3.x & mask) ? 1 : 0;
    int yi = (index3.y & mask) ? 1 : 0;
    int zi = (index3.z & mask) ? 1 : 0;        
    int index = (xi << 2) + (yi << 1) + zi;
    if(index == 0) {
      index3.w = index3.z;
      index3.z = index3.y;
      index3.y = index3.w;
    } else if(index == 1 || index == 5) {
      index3.w = index3.x;
      index3.x = index3.y;
      index3.y = index3.w;
    } else if(index == 4 || index == 6) {
      index3.x = (index3.x) ^ (-1);
      index3.z = (index3.z) ^ (-1);
    } else if(index == 7 || index == 3) {
      index3.w = (index3.x) ^ (-1);         
      index3.x = (index3.y) ^ (-1);
      index3.y = index3.w;
    } else {
      index3.w = (index3.z) ^ (-1);         
      index3.z = (index3.y) ^ (-1);
      index3.y = index3.w;          
    }   
    key = (key << 3) + C[index];
    if(i == 19) {
      key4.y = key;
      key = 0;
    }
    if(i == 9) {
      key4.x = key;
      key = 0;
    }
  }
  key4.z = key;
  return key4;
}

static __device__ uint4 getMask(int level) {
  int mask_levels = 3 * (MAXLEVELS - level);
  uint4 mask = {0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF};
  if (mask_levels > 60) {
    mask.z = 0;
    mask.y = 0;
    mask.x = (mask.x >> (mask_levels - 60)) << (mask_levels - 60);
  } else if (mask_levels > 30) {
    mask.z = 0;
    mask.y = (mask.y >> (mask_levels - 30)) << (mask_levels - 30);
  } else {
    mask.z = (mask.z >> mask_levels) << mask_levels;
  }
  return mask;
}

static __device__ int compareKey(uint4 a, uint4 b) {
  if      (a.x < b.x) return -1;
  else if (a.x > b.x) return +1;
  else {
    if       (a.y < b.y) return -1;
    else  if (a.y > b.y) return +1;
    else {
      if       (a.z < b.z) return -1;
      else  if (a.z > b.z) return +1;
      return 0;
    }
  }
}

//Binary search of the key within certain bounds (cij.x, cij.y)
static __device__ int findKey(uint4 key, uint2 cij, uint4 *keys) {
  int l = cij.x;
  int r = cij.y - 1;
  while (r - l > 1) {
    int m = (r + l) >> 1;
    int cmp = compareKey(keys[m], key);
    if (cmp == -1)
      l = m;
    else 
      r = m;
  }
  if (compareKey(keys[l], key) >= 0) return l;
  return r;
}

extern "C" __global__ void boundaryReduction(const int numBodies,
                                             float4 *bodyPos,
                                             float3 *output_xmin,
                                             float3 *output_xmax)
{
  float3 xmin = make_float3(+1e10f, +1e10f, +1e10f);
  float3 xmax = make_float3(-1e10f, -1e10f, -1e10f);
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  const uint stride = blockDim.x * gridDim.x;
  while (idx < numBodies) {
    float4 pos = bodyPos[idx];
    pairMinMax(xmin, xmax, pos, pos);
    idx += stride;
  }
  sharedMinMax(xmin,xmax);

  if( threadIdx.x == 0 ) {
    output_xmin[blockIdx.x].x = xmin.x;
    output_xmin[blockIdx.x].y = xmin.y;
    output_xmin[blockIdx.x].z = xmin.z;
    output_xmax[blockIdx.x].x = xmax.x;
    output_xmax[blockIdx.x].y = xmax.y;
    output_xmax[blockIdx.x].z = xmax.z;
  }
}

extern "C" __global__ void getKeyKernel(int numBodies,
                                        float4 corner,
                                        float4 *bodyPos,
                                        uint4 *bodyKeys) {
  const uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= numBodies) return;
  float4 pos = bodyPos[idx];
  int4 index3;
  index3.x = (int)roundf(__fdividef(pos.x - corner.x, corner.w));
  index3.y = (int)roundf(__fdividef(pos.y - corner.y, corner.w));
  index3.z = (int)roundf(__fdividef(pos.z - corner.z, corner.w));
  uint4 key = getKey(index3);
  key.w = idx;
  bodyKeys[idx] = key;
}

extern "C" __global__ void getValidRange(int numBodies,
                                         int level,
                                         uint4 *bodyKeys,
                                         uint *validRange,
                                         const uint *workToDo) {
  if (*workToDo == 0) return;
  const uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= numBodies) return;
  const uint4 key_F = {0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF};
  uint4 mask = getMask(level);
  uint4 key_c = bodyKeys[idx];
  uint4 key_m;
  if( idx == 0 )
    key_m = key_F;
  else
    key_m = bodyKeys[idx-1];

  uint4 key_p;
  if( idx == numBodies-1 )
    key_p = key_F;
  else
    key_p = bodyKeys[idx+1];

  int valid0 = 0;
  int valid1 = 0;
  if (compareKey(key_c, key_F) != 0) {
    key_c.x = key_c.x & mask.x;
    key_c.y = key_c.y & mask.y;
    key_c.z = key_c.z & mask.z;
    key_p.x = key_p.x & mask.x;
    key_p.y = key_p.y & mask.y;
    key_p.z = key_p.z & mask.z;
    key_m.x = key_m.x & mask.x;
    key_m.y = key_m.y & mask.y;
    key_m.z = key_m.z & mask.z;
    valid0 = abs(compareKey(key_c, key_m));
    valid1 = abs(compareKey(key_c, key_p));
  }
  validRange[idx*2]   = idx | ((valid0) << 31);
  validRange[idx*2+1] = idx | ((valid1) << 31);
}

extern "C" __global__ void buildNodes(
                             uint level,
                             uint *workToDo,
                             uint *maxLevel,
                             uint2 *levelRange,
                             uint *bodyOffset,
                             uint4 *bodyKeys,
                             uint4 *cellKeys,
                             uint2 *bodyRange) {
  if( *workToDo == 0 ) return;
  uint idx  = blockIdx.x * blockDim.x + threadIdx.x;
  const uint stride = gridDim.x * blockDim.x;
  uint n = (*workToDo) / 2;
  uint offset;
  if( level == 0 )
    offset = 0;
  else
    offset = levelRange[level-1].y;

  while( idx < n ) {
    uint begin = bodyOffset[idx*2];
    uint end = bodyOffset[idx*2+1]+1;
    uint4 key  = bodyKeys[begin];
    uint4 mask = getMask(level);
    key = make_uint4(key.x & mask.x, key.y & mask.y, key.z & mask.z, level); 
    bodyRange[offset+idx] = make_uint2(begin, end);
    cellKeys  [offset+idx] = key;
    if( end - begin <= NLEAF )
      for( int i=begin; i<end; i++ )
        bodyKeys[i] = make_uint4(0xFFFFFFFF,0xFFFFFFFF,0xFFFFFFFF,0xFFFFFFFF);
    idx += stride;
  }

  if( threadIdx.x == 0 && blockIdx.x == 0 ) {
    levelRange[level] = make_uint2(offset, offset + n);
    *maxLevel = level;
  }
}

extern "C" __global__ void linkNodes(int numSources,
                                     float4 corner,
                                     uint2 *bodyRange,
                                     uint4 *cellKeys,
                                     uint *childRange,
                                     uint2 *levelRange,
                                     uint* validRange) {
  const uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= numSources) return;
  uint4 key = cellKeys[idx];
  uint level = key.w;
  uint begin = bodyRange[idx].x;
  uint end   = bodyRange[idx].y;

  uint4 mask = getMask(level-1);
  key = make_uint4(key.x & mask.x, key.y & mask.y,  key.z & mask.z, 0);
  if(idx > 0) {
    int ci = findKey(key,levelRange[level-1],cellKeys);
    atomicAdd(&childRange[ci], (1 << 28));
  }

  key = cellKeys[idx];
  mask = getMask(level);
  key = make_uint4(key.x & mask.x, key.y & mask.y, key.z & mask.z, 0);
  int cj = findKey(key,levelRange[level+1],cellKeys);
  atomicOr(&childRange[idx], cj);

  uint valid = idx;
  if( end - begin <= NLEAF )
    valid = idx | (uint)(1 << 31);
  validRange[idx] = valid;
}

extern "C" __global__ void getLevelRange(const int numSources,
                                         const int numLeafs,
                                         uint *cellIndex,
                                         uint4 *cellKeys,
                                         uint* validRange) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x + numLeafs;
  if (idx >= numSources) return;
  const int cellIdx = cellIndex[idx];
  int level_c, level_m, level_p;
  level_c = cellKeys[cellIndex[idx]].w;
  if( idx+1 < numSources )
    level_p = cellKeys[cellIndex[idx+1]].w;
  else
    level_p = MAXLEVELS+5;
  if(cellIdx == 0)
    level_m = -1;    
  else
    level_m = cellKeys[cellIndex[idx-1]].w;
  validRange[(idx-numLeafs)*2]   = idx | (level_c != level_m) << 31;
  validRange[(idx-numLeafs)*2+1] = idx | (level_c != level_p) << 31;
}

extern "C" __global__ void getTargetRange(int numBodies,
                                          uint *validRange,
                                          uint *levelOffset,
                                          int treeDepth) {
  const uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= numBodies) return;
  __shared__ int shmem[128];
  if(blockIdx.x == 0) {
    if(threadIdx.x < (MAXLEVELS*2))
      shmem[threadIdx.x] = levelOffset[threadIdx.x];
    __syncthreads();
    if(threadIdx.x < MAXLEVELS) {
      levelOffset[threadIdx.x]  = shmem[threadIdx.x*2];
      if(threadIdx.x == treeDepth-1)
        levelOffset[threadIdx.x] = shmem[threadIdx.x*2-1]+1;
    }
  }
  int validBegin = ((idx     % NCRIT) == 0);
  int validEnd   = (((idx+1) % NCRIT) == 0);
  if(idx+1 == numBodies) validEnd = 1;
  validRange[2*idx + 0] = (idx)   | (uint)(validBegin << 31);
  validRange[2*idx + 1] = (idx+1) | (uint)(validEnd   << 31);    
}

extern "C" __global__ void storeTargetRange(int numTargets,
                                            uint *validRange,
                                            uint2 *targetRange) {
  if(blockIdx.x >= numTargets) return;
  if(threadIdx.x == 0) {
    int begin = validRange[2*blockIdx.x];
    int end   = validRange[2*blockIdx.x+1];
    targetRange[blockIdx.x] = make_uint2(begin,end);
  }
}

extern "C" __global__ void reorder(const int size, uint4 *index, float4 *input, float4* output) {
  const uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size) return;
  int newIndex = index[idx].w;
  output[idx] = input[newIndex];
}

extern "C" __global__ void P2M(const int numLeafs,
                               uint *cellIndex,
                               uint2 *bodyRange,
                               float4 *bodyPos,
                               float4 *cellPos,
                               float4 *cellXmin,
                               float4 *cellXmax,
                               float  *multipole) {
  const uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= numLeafs) return;
  int cellIdx = cellIndex[idx];
  const uint begin = bodyRange[cellIdx].x;
  const uint end   = bodyRange[cellIdx].y;
  float4 mon = {0.0f, 0.0f, 0.0f, 0.0f};
  float3 xmin, xmax;
  xmin = make_float3(+1e10f, +1e10f, +1e10f);
  xmax = make_float3(-1e10f, -1e10f, -1e10f);
  for( int i=begin; i<end; i++ ) {
    float4 pos = bodyPos[i];
    mon.w += pos.w;
    mon.x += pos.w * pos.x;
    mon.y += pos.w * pos.y;
    mon.z += pos.w * pos.z;
    pairMinMax(xmin, xmax, pos, pos);
  }
  float im = 1.0/mon.w;
  if(mon.w == 0) im = 0;
  mon.x *= im;
  mon.y *= im;
  mon.z *= im;
  cellPos[cellIdx] = make_float4(mon.x, mon.y, mon.z, mon.w);
  cellXmin[cellIdx] = make_float4(xmin.x, xmin.y, xmin.z, 0.0f);
  cellXmax[cellIdx] = make_float4(xmax.x, xmax.y, xmax.z, 1.0f);
  return;
}

extern "C" __global__ void M2M(const int level,
                               uint *cellIndex,
                               uint *levelOffset,
                               uint *childRange,
                               float4 *cellPos,
                               float4 *cellXmin,
                               float4 *cellXmax,
                               float  *multipole) {
  const uint idx = blockIdx.x * blockDim.x + threadIdx.x + levelOffset[level-1];
  if(idx >= levelOffset[level]) return;
  const int cellIdx = cellIndex[idx];
  const uint begin = childRange[cellIdx] & 0x0FFFFFFF;
  const uint nchild = ((childRange[cellIdx] & 0xF0000000) >> 28);
  const uint end = begin + nchild;
  childRange[cellIdx] = begin | ((nchild-1) << LEAFBIT);
  float4 mon = {0.0f, 0.0f, 0.0f, 0.0f};
  float3 xmin = make_float3(+1e10f, +1e10f, +1e10f);
  float3 xmax = make_float3(-1e10f, -1e10f, -1e10f);
  for( int i=begin; i<end; i++ ) {
    float4 pos = cellPos[i];
    mon.w += pos.w;
    mon.x += pos.w * pos.x;
    mon.y += pos.w * pos.y;
    mon.z += pos.w * pos.z;
    pairMinMax(xmin, xmax, cellXmin[i], cellXmax[i]);
  }
  float im = 1.0 / mon.w;
  if(mon.w == 0) im = 0;
  mon.x *= im;
  mon.y *= im;
  mon.z *= im;
  cellPos[cellIdx] = make_float4(mon.x, mon.y, mon.z, mon.w);
  cellXmin[cellIdx] = make_float4(xmin.x, xmin.y, xmin.z, 0.0f);
  cellXmax[cellIdx] = make_float4(xmax.x, xmax.y, xmax.z, 0.0f);
  return;
}

extern "C" __global__ void rescale(const int node_count,
                                   float4 *cellPos,
                                   float4 *cellXmin,
                                   float4 *cellXmax,
                                   uint  *childRange,
                                   float *openingAngle,
                                   uint2 *bodyRange){
  const uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx >= node_count) return;
  float4 mon = cellPos[idx];
  float4 xmin = cellXmin[idx];
  float4 xmax = cellXmax[idx];
  float3 boxCenter = make_float3(0.5*(xmin.x + xmax.x),
                                 0.5*(xmin.y + xmax.y),
                                 0.5*(xmin.z + xmax.z));
  float3 boxSize = make_float3(fmaxf(fabs(boxCenter.x-xmin.x), fabs(boxCenter.x-xmax.x)),
                               fmaxf(fabs(boxCenter.y-xmin.y), fabs(boxCenter.y-xmax.y)),
                               fmaxf(fabs(boxCenter.z-xmin.z), fabs(boxCenter.z-xmax.z)));
  float3 dX = make_float3((boxCenter.x - mon.x), (boxCenter.y - mon.y), (boxCenter.z - mon.z));
  float R = sqrt((dX.x*dX.x) + (dX.y*dX.y) + (dX.z*dX.z));
  if(fabs(mon.w) < EPS) R = 0;

  float length = 2 * fmaxf(boxSize.x, fmaxf(boxSize.y, boxSize.z));
  if(length < EPS) length = EPS;
  float cellOp = length / THETA + R;
  cellOp = cellOp * cellOp;
  uint pfirst = bodyRange[idx].x;
  uint nchild = bodyRange[idx].y - pfirst;
  bool leaf = (xmax.w > 0);

  if( leaf ) {
    cellOp = -cellOp;
    pfirst = pfirst | ((nchild-1) << LEAFBIT);
    childRange[idx] = pfirst;
  }
  openingAngle[idx] = cellOp;
  return;
}

extern "C" __global__ void setTargets(const int numTargets,
                                      float4 *bodyPos,
                                      int2   *targetRange,
                                      float4 *targetCenterInfo,
                                      float4 *targetSizeInfo){
  if(blockIdx.x >= numTargets) return;
  float3 xmin = make_float3(+1e10f, +1e10f, +1e10f);
  float3 xmax = make_float3(-1e10f, -1e10f, -1e10f);
  int begin = targetRange[blockIdx.x].x;
  int end   = targetRange[blockIdx.x].y;
  int idx = begin + threadIdx.x;
  if( idx < end ) {
    float4 pos = bodyPos[idx];
    pairMinMax(xmin, xmax, pos, pos);
  }
  sharedMinMax(xmin,xmax);
  if( threadIdx.x == 0 ) {
    float3 targetCenter = make_float3(0.5*(xmin.x + xmax.x),
                                     0.5*(xmin.y + xmax.y),
                                     0.5*(xmin.z + xmax.z));
    float3 targetSize = make_float3(fmaxf(fabs(targetCenter.x-xmin.x), fabs(targetCenter.x-xmax.x)),
                                   fmaxf(fabs(targetCenter.y-xmin.y), fabs(targetCenter.y-xmax.y)),
                                   fmaxf(fabs(targetCenter.z-xmin.z), fabs(targetCenter.z-xmax.z)));
    int nchild = end-begin;
    begin = begin | (nchild-1) << CRITBIT;
    targetSizeInfo[blockIdx.x].x = targetSize.x;
    targetSizeInfo[blockIdx.x].y = targetSize.y;
    targetSizeInfo[blockIdx.x].z = targetSize.z;
    targetSizeInfo[blockIdx.x].w = __int_as_float(begin);
    float length = max(targetSize.x, max(targetSize.y, targetSize.z));
    targetCenterInfo[blockIdx.x].x = targetCenter.x;
    targetCenterInfo[blockIdx.x].y = targetCenter.y;
    targetCenterInfo[blockIdx.x].z = targetCenter.z;
    targetCenterInfo[blockIdx.x].w = length;
  }
}

void octree::getBoundaries() {
  boundaryReduction<<<64,NCRIT>>>(numBodies,bodyPos.devc(),XMIN.devc(),XMAX.devc());
  XMIN.d2h();
  XMAX.d2h();
  float4 xmin = make_float4(+1e10, +1e10, +1e10, +1e10);
  float4 xmax = make_float4(-1e10, -1e10, -1e10, -1e10);
  for (int i = 0; i < 64; i++) {
    xmin.x = std::min(xmin.x, XMIN[i].x);
    xmin.y = std::min(xmin.y, XMIN[i].y);
    xmin.z = std::min(xmin.z, XMIN[i].z);
    xmax.x = std::max(xmax.x, XMAX[i].x);
    xmax.y = std::max(xmax.y, XMAX[i].y);
    xmax.z = std::max(xmax.z, XMAX[i].z);
  }
  float size = 1.001f*std::max(xmax.z - xmin.z,
                      std::max(xmax.y - xmin.y, xmax.x - xmin.x));
  corner = make_float4(0.5f*(xmin.x + xmax.x) - 0.5f*size,
                       0.5f*(xmin.y + xmax.y) - 0.5f*size,
                       0.5f*(xmin.z + xmax.z) - 0.5f*size,
                       size / (1 << MAXLEVELS));
}

void octree::getKeys() {
  int threads = 128;
  int blocks = ALIGN(numBodies,threads);
  getKeyKernel<<<blocks,threads>>>(numBodies,corner,bodyPos.devc(),uint4buffer);
}

void octree::sortKeys() {
  sorter->sort(uint4buffer,bodyKeys,numBodies);
}

void octree::sortBodies() {
  int threads = 512;
  int blocks = ALIGN(numBodies,threads);
  reorder<<<blocks,threads>>>(numBodies,bodyKeys.devc(),bodyPos.devc(),float4buffer);
  CU_SAFE_CALL(hipMemcpy(bodyPos.devc(),float4buffer,numBodies*sizeof(float4),hipMemcpyDeviceToDevice));
}

void octree::buildTree() {
  cudaVec<uint> maxLevel;
  maxLevel.alloc(1);
  validRange.zeros();
  levelRange.zeros();
  workToDo.ones();
  int threads = 128;
  int blocks = ALIGN(numBodies,threads);
  for( int level=0; level<MAXLEVELS; level++ ) {
    getValidRange<<<blocks,threads>>>(numBodies,level,bodyKeys.devc(),validRange.devc(),workToDo.devc());
    gpuCompact(validRange,compactRange,2*numBodies);
    buildNodes<<<64,threads>>>(level,workToDo.devc(),maxLevel.devc(),levelRange.devc(),compactRange.devc(),bodyKeys.devc(),cellKeys.devc(),bodyRange.devc());
  }
  maxLevel.d2h();
  numLevels = maxLevel[0];
  levelRange.d2h();
  numSources = levelRange[numLevels].y;
}

void octree::linkTree() {
  // cellIndex
  childRange.zeros();
  int threads = 128;
  int blocks = ALIGN(numSources,threads);
  linkNodes<<<blocks,threads>>>(numSources,corner,bodyRange.devc(),cellKeys.devc(),childRange.devc(),levelRange.devc(),validRange.devc());
  cellIndex.alloc(numSources);
  workToDo.ones();
  gpuSplit(validRange, cellIndex, numSources);
  workToDo.d2h();
  numLeafs = workToDo[0];
  // levelOffset
  validRange.zeros();
  blocks = ALIGN(numSources-numLeafs,threads);
  getLevelRange<<<blocks,threads>>>(numSources,numLeafs,cellIndex.devc(),cellKeys.devc(),validRange.devc());
  gpuCompact(validRange, levelOffset, 2*(numSources-numLeafs));
  // targetRange
  validRange.zeros();
  blocks = ALIGN(numBodies,threads);
  getTargetRange<<<blocks,threads>>>(numBodies,validRange.devc(),levelOffset.devc(),numLevels+1);
  gpuCompact(validRange, compactRange, numBodies*2);
  workToDo.d2h();
  numTargets = workToDo[0] / 2;
  targetRange.alloc(numTargets);
  storeTargetRange<<<numTargets,NCRIT>>>(numTargets,compactRange.devc(),targetRange.devc());
}

void octree::allocateTreePropMemory()
{
  cellPos.alloc(numSources);
  multipole.alloc(MTERM*numSources);
  targetSizeInfo.alloc(numSources);
  openingAngle.alloc(numSources);
  targetCenterInfo.alloc(numSources);
}

void octree::upward() {
  cudaVec<float4> cellXmin;
  cudaVec<float4> cellXmax;
  cellXmin.alloc(numSources);
  cellXmax.alloc(numSources);

  int threads = 128;
  int blocks = ALIGN(numLeafs,threads);
  P2M<<<blocks,threads>>>(numLeafs,cellIndex.devc(),bodyRange.devc(),bodyPos.devc(),
                                       cellPos.devc(),cellXmin.devc(),cellXmax.devc(),multipole.devc());

  levelOffset.d2h();
  for( int level=numLevels; level>=1; level-- ) {
    int totalOnThisLevel = levelOffset[level] - levelOffset[level-1];
    blocks = ALIGN(totalOnThisLevel,threads);
    M2M<<<blocks,threads>>>(level,cellIndex.devc(),levelOffset.devc(),childRange.devc(),
                                         cellPos.devc(),cellXmin.devc(),cellXmax.devc(),multipole.devc());
  }

  blocks = ALIGN(numSources,threads);
  rescale<<<blocks,threads>>>(numSources,cellPos.devc(),cellXmin.devc(),cellXmax.devc(),childRange.devc(),openingAngle.devc(),bodyRange.devc());
  setTargets<<<numTargets,NCRIT>>>(numTargets,bodyPos.devc(),(int2*)targetRange.devc(),targetCenterInfo.devc(),targetSizeInfo.devc());
}
