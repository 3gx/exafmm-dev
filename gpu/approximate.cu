#include "hip/hip_runtime.h"
#include "octree.h"
#define laneId (threadIdx.x & (WARP_SIZE - 1))
#define warpId (threadIdx.x >> WARP_SIZE2)
#define IF(x) (-(int)(x))
#define ABS(x) ((int(x) < 0 ) ? -(x) : (x))

__device__ __forceinline__ int inclusiveScanInt(int* prefix, int value) 
{
  prefix[laneId] = value;
  for (int i = 0; i < WARP_SIZE2; i++) {
    const int offset = 1 << i;
    const int laneOffset = ABS(laneId-offset);
    prefix[laneId] += prefix[laneOffset] & IF(laneId >= offset);
  }
  return prefix[WARP_SIZE-1];
}

__device__ __forceinline__ int lanemask_lt()
{
  int mask;
  asm("mov.u32 %0, %lanemask_lt;" : "=r" (mask));
  return mask;
}

__device__ int exclusiveScanBit(const bool flag)
{
  const uint flags = __ballot(flag);
  return __popc(flags & lanemask_lt());
}

__device__ int reduceBit(const bool flag)
{
  const uint flags = __ballot(flag);
  return __popc(flags);
}

__device__ __forceinline__ int lanemask_le()
{
  int mask;
  asm("mov.u32 %0, %lanemask_le;" : "=r" (mask));
  return mask;
}

__device__ __forceinline__ int inclusive_segscan_warp(
    int *shmem, const int packed_value, int &dist_block, int &nseg)
{
  const int  flag = packed_value < 0;
  const int  mask = IF(flag);
  const int value = (mask & (-1-packed_value)) + (~mask & 1);
  const int flags = __ballot(flag);

  nseg += __popc(flags) ;
  dist_block = __clz(__brev(flags));

  const int distance = min(__clz(flags & lanemask_le()) + laneId - 31, laneId);
  shmem[laneId] = value;
  for( int i=0; i<WARP_SIZE2; i++ ) {
    const int offset = 1 << i;
    const int laneOffset = ABS(laneId-offset);
    shmem[laneId] += shmem[laneOffset] & IF(offset <= distance);
  }
  return shmem[WARP_SIZE - 1];
}

__device__ __forceinline__ int inclusive_segscan_array(int *shmem_in, const int N)
{
  int dist, nseg = 0;
  int y = inclusive_segscan_warp(shmem_in, shmem_in[laneId], dist, nseg);
  for( int p=WARP_SIZE; p<N; p+=WARP_SIZE ) {
    int *shmem = shmem_in + p;
    int y1 = inclusive_segscan_warp(shmem, shmem[laneId], dist, nseg);
    shmem[laneId] += y & IF(laneId < dist);
    y = y1;
  }
  return nseg;
}

__device__ __forceinline__ int ACCESS(const int i) {
  return (i & (LMEM_STACK_SIZE - 1)) * blockDim.x + threadIdx.x;
}

texture<uint, 1, hipReadModeElementType> texNodeChild;
texture<float, 1, hipReadModeElementType> texOpening;
texture<float4, 1, hipReadModeElementType> texMultipole;
texture<float4, 1, hipReadModeElementType> texBody;

__device__ __forceinline__ void P2P(
    float4 &acc,  const float4 pos,
    const float4 posj) {
  const float3 dr = make_float3(posj.x - pos.x, posj.y - pos.y, posj.z - pos.z);
  const float r2     = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z + EPS2;
  const float rinv   = rsqrtf(r2);
  const float rinv2  = rinv*rinv;
  const float mrinv  = posj.w * rinv;
  const float mrinv3 = mrinv * rinv2;
  acc.w -= mrinv;
  acc.x += mrinv3 * dr.x;
  acc.y += mrinv3 * dr.y;
  acc.z += mrinv3 * dr.z;
}

__device__ bool applyMAC(
    const float4 sourceCenter, 
    const float4 groupCenter, 
    const float4 groupSize) {
  float3 dr = make_float3(fabsf(groupCenter.x - sourceCenter.x) - (groupSize.x),
                          fabsf(groupCenter.y - sourceCenter.y) - (groupSize.y),
                          fabsf(groupCenter.z - sourceCenter.z) - (groupSize.z));
  dr.x += fabsf(dr.x); dr.x *= 0.5f;
  dr.y += fabsf(dr.y); dr.y *= 0.5f;
  dr.z += fabsf(dr.z); dr.z *= 0.5f;
  const float ds2 = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z;
  return ds2 <= fabsf(sourceCenter.w);
}

__device__ void traverse(
    float4 &pos_i,
    float4 &acc_i,
    float4 targetCenter,
    float4 targetSize,
    uint2 rootRange,
    int *shmem,
    int *lmem) {
  const int stackSize = LMEM_STACK_SIZE << NTHREAD2;
  int *approxNodes = lmem + stackSize + 2 * WARP_SIZE * warpId;
  int *numDirect = shmem;
  int *stackShrd = numDirect + WARP_SIZE;
  int *directNodes = stackShrd + WARP_SIZE;
  float4 *pos_j = (float4*)&directNodes[3*WARP_SIZE];
  int *prefix = (int*)&pos_j[WARP_SIZE];

  // stack
  int *stackGlob = lmem;
  // begin tree-walk
  int warpOffsetApprox = 0;
  int warpOffsetDirect = 0;
  for( int root=rootRange.x; root<rootRange.y; root+=WARP_SIZE ) {
    int numNodes = min(rootRange.y-root, WARP_SIZE);
    int beginStack = 0;
    int endStack = 1;
    stackGlob[threadIdx.x] = root + laneId;
    // walk each level
    while( numNodes > 0 ) {
      int numNodesNew = 0;
      int warpOffsetSplit = 0;
      int numStack = endStack;
      // walk a level
      for( int iStack=beginStack; iStack<endStack; iStack++ ) {
        bool valid = laneId < numNodes;
        int node = stackGlob[ACCESS(iStack)] & IF(valid);
        numNodes -= WARP_SIZE;
        float opening = tex1Dfetch(texOpening, node);
        uint sourceData = tex1Dfetch(texNodeChild, node);
        float4 sourceCenter = tex1Dfetch(texMultipole, node);
        sourceCenter.w = opening;
        bool split = applyMAC(sourceCenter, targetCenter, targetSize);
        bool leaf = opening <= 0;
        bool flag = split && !leaf && valid;
        int child = sourceData & 0x0FFFFFFF;
        int numChild = ((sourceData & 0xF0000000) >> 28) & IF(flag);
        int sumChild = inclusiveScanInt(prefix, numChild);
        int laneOffset = prefix[laneId];
        laneOffset += warpOffsetSplit - numChild;
        for( int i=0; i<numChild; i++ )
          stackShrd[laneOffset+i] = child+i;
        warpOffsetSplit += sumChild;
        while( warpOffsetSplit >= WARP_SIZE ) {
          warpOffsetSplit -= WARP_SIZE;
          stackGlob[ACCESS(numStack)] = stackShrd[warpOffsetSplit+laneId];
          numStack++;
          numNodesNew += WARP_SIZE;
          if( (numStack - iStack) > LMEM_STACK_SIZE ) return;
        }
#if 1   // APPROX
        flag = !split && valid;
        laneOffset = exclusiveScanBit(flag);
        if( flag ) approxNodes[warpOffsetApprox+laneOffset] = node;
        warpOffsetApprox += reduceBit(flag);
        if( warpOffsetApprox >= WARP_SIZE ) {
          warpOffsetApprox -= WARP_SIZE;
          node = approxNodes[warpOffsetApprox+laneId];
          pos_j[laneId] = tex1Dfetch(texMultipole, node);
          for( int i=0; i<WARP_SIZE; i++ )
            P2P(acc_i, pos_i, pos_j[i]);
        }
#endif
#if 1   // DIRECT
        flag = split && leaf && valid;
        const int jbody = sourceData & BODYMASK;
        int numBodies = (((sourceData & INVBMASK) >> LEAFBIT)+1) & IF(flag);
        directNodes[laneId] = numDirect[laneId];

        int sumBodies = inclusiveScanInt(prefix, numBodies);
        laneOffset = prefix[laneId];
        if( flag ) prefix[exclusiveScanBit(flag)] = laneId;
        numDirect[laneId] = laneOffset;
        laneOffset -= numBodies;
        int numFinished = 0;
        while( sumBodies > 0 ) {
          numBodies = min(sumBodies, 3*WARP_SIZE-warpOffsetDirect);
          for( int i=warpOffsetDirect; i<warpOffsetDirect+numBodies; i+=WARP_SIZE )
            directNodes[i+laneId] = 0;
          if( flag && (numDirect[laneId] <= numBodies) && (laneOffset >= 0) )
            directNodes[warpOffsetDirect+laneOffset] = -1-jbody;
          numFinished += inclusive_segscan_array(&directNodes[warpOffsetDirect], numBodies);
          numBodies = numDirect[prefix[numFinished-1]];
          sumBodies -= numBodies;
          numDirect[laneId] -= numBodies;
          laneOffset -= numBodies;
          warpOffsetDirect += numBodies;
          while( warpOffsetDirect >= WARP_SIZE ) {
            warpOffsetDirect -= WARP_SIZE;
            pos_j[laneId] = tex1Dfetch(texBody,directNodes[warpOffsetDirect+laneId]);
            for( int i=0; i<WARP_SIZE; i++ )
              P2P(acc_i, pos_i, pos_j[i]);
          }
        }
        numDirect[laneId] = directNodes[laneId];
#endif
      }

      if( warpOffsetSplit > 0 ) { 
        stackGlob[ACCESS(numStack)] = stackShrd[laneId];
        numStack++; 
        numNodesNew += warpOffsetSplit;
      }
      numNodes = numNodesNew;
      beginStack = endStack;
      endStack = numStack;
    }
  }

  if( warpOffsetApprox > 0 ) {
    if( laneId < warpOffsetApprox )  {
      const int node = approxNodes[laneId];
      pos_j[laneId] = tex1Dfetch(texMultipole, node);
    } else {
      pos_j[laneId] = make_float4(1.0e10f, 1.0e10f, 1.0e10f, 0.0f);
    }
    for( int i=0; i<WARP_SIZE; i++ )
      P2P(acc_i, pos_i, pos_j[i]);
  }

  if( warpOffsetDirect > 0 ) {
    if( laneId < warpOffsetDirect ) {
      const float4 posj = tex1Dfetch(texBody,numDirect[laneId]);
      pos_j[laneId] = posj;
    } else {
      pos_j[laneId] = make_float4(1.0e10f, 1.0e10f, 1.0e10f, 0.0f);
    }
    for( int i=0; i<WARP_SIZE; i++ ) 
      P2P(acc_i, pos_i, pos_j[i]);
  }
}

extern "C" __global__ void
  traverseKernel(
      const int numGroups,
      uint2 *levelRange,
      float4 *acc,
      float4 *groupSizeInfo,
      float4 *groupCenterInfo,
      int    *MEM_BUF,
      uint   *workToDo) {
  __shared__ int wid[4];
  __shared__ int shmem_pool[10*NTHREAD];
  int *shmem = shmem_pool+10*WARP_SIZE*warpId;
  int *lmem = &MEM_BUF[blockIdx.x*(LMEM_STACK_SIZE*NTHREAD+2*NTHREAD)];
  while(true) {
    if( laneId == 0 )
      wid[warpId] = atomicAdd(workToDo,1);
    if( wid[warpId] >= numGroups ) return;
    float4 groupSize = groupSizeInfo[wid[warpId]];
    const int groupData = __float_as_int(groupSize.w);
    const uint begin = groupData & CRITMASK;
    const uint numGroup = ((groupData & INVCMASK) >> CRITBIT) + 1;
    float4 groupCenter = groupCenterInfo[wid[warpId]];
    uint body_i = begin + laneId % numGroup;
    float4 pos_i = tex1Dfetch(texBody,body_i);
    float4 acc_i = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

    traverse(pos_i, acc_i, groupCenter, groupSize, levelRange[2], shmem, lmem);
    if( laneId < numGroup )
      acc[body_i] = acc_i;
  }
}

extern "C" __global__ void directKernel(float4 *bodyPos, float4 *bodyAcc, const int N) {
  uint idx = min(blockIdx.x * blockDim.x + threadIdx.x, N-1);
  float4 pos_i = bodyPos[idx];
  float4 acc_i = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
  __shared__ float4 shmem[NTHREAD];
  float4 *pos_j = shmem + WARP_SIZE * warpId;
  const int numWarp = ALIGN(N, WARP_SIZE);
  for( int jwarp=0; jwarp<numWarp; jwarp++ ) {
    int jGlob = jwarp*WARP_SIZE+laneId;
    pos_j[laneId] = bodyPos[min(jGlob,N-1)];
    pos_j[laneId].w *= jGlob < N;
    for( int i=0; i<WARP_SIZE; i++ )
      P2P(acc_i, pos_i, pos_j[i]);
  }
  bodyAcc[idx] = acc_i;
}

void octree::traverse() {
  nodeChild.tex("texNodeChild");
  openingAngle.tex("texOpening");
  multipole.tex("texMultipole");
  bodyPos.tex("texBody");
  workToDo.zeros();
  traverseKernel<<<NBLOCK,NTHREAD,0,execStream>>>(
    numGroups,
    levelRange.devc(),
    bodyAcc.devc(),
    groupSizeInfo.devc(),
    groupCenterInfo.devc(),
    (int*)generalBuffer1.devc(),
    workToDo.devc()
  );
}

void octree::iterate() {
  CU_SAFE_CALL(hipStreamCreate(&execStream));
  double t1 = get_time();
  getBoundaries();
  CU_SAFE_CALL(hipStreamSynchronize(execStream));
  printf("BOUND : %lf\n",get_time() - t1);;
  t1 = get_time();
  getKeys();
  CU_SAFE_CALL(hipStreamSynchronize(execStream));
  printf("INDEX : %lf\n",get_time() - t1);;
  t1 = get_time();
  sortKeys();
  CU_SAFE_CALL(hipStreamSynchronize(execStream));
  printf("KEYS  : %lf\n",get_time() - t1);;
  t1 = get_time();
  sortBodies();
  CU_SAFE_CALL(hipStreamSynchronize(execStream));
  printf("BODIES: %lf\n",get_time() - t1);;
  t1 = get_time();
  buildTree();
  CU_SAFE_CALL(hipStreamSynchronize(execStream));
  printf("BUILD : %lf\n",get_time() - t1);;
  t1 = get_time();
  allocateTreePropMemory();
  CU_SAFE_CALL(hipStreamSynchronize(execStream));
  printf("ALLOC : %lf\n",get_time() - t1);;
  t1 = get_time();
  linkTree();
  CU_SAFE_CALL(hipStreamSynchronize(execStream));
  printf("LINK  : %lf\n",get_time() - t1);;
  t1 = get_time();
  upward();
  CU_SAFE_CALL(hipStreamSynchronize(execStream));
  printf("UPWARD: %lf\n",get_time() - t1);;
  t1 = get_time();
  traverse();
  CU_SAFE_CALL(hipStreamSynchronize(execStream));
  printf("FMM   : %lf\n",get_time() - t1);;
}

void octree::direct() {
  int blocks = ALIGN(numBodies/100, NTHREAD);
  directKernel<<<blocks,NTHREAD,0,execStream>>>(bodyPos.devc(),bodyAcc2.devc(),numBodies);
  CU_SAFE_CALL(hipStreamSynchronize(execStream));
  CU_SAFE_CALL(hipStreamDestroy(execStream));
}
