#include "hip/hip_runtime.h"
#include "octree.h"

int main() {
  for( int it=0; it<25; it++ ) {
    uint numBodies = uint(pow(10,(it+24)/8.0));
    uint numTarget = numBodies / 100;
    octree *tree = new octree(numBodies);
    printf("N     : %d\n",numBodies);
    for( uint i=0; i<numBodies; i++ ) {
      tree->bodyPos[i].w  = 1. / numBodies;
      tree->bodyPos[i].x  = drand48();
      tree->bodyPos[i].y  = drand48();
      tree->bodyPos[i].z  = drand48();
    }
    tree->bodyPos.h2d();
    tree->iterate(); 
    double tic = tree->get_time();
    tree->direct(numTarget,numBodies);
    double toc = tree->get_time();
    tree->bodyAcc.d2h();
    tree->bodyAcc2.d2h();
    float diff1 = 0, norm1 = 0, diff2 = 0, norm2 = 0;
    for( uint i=0; i<numTarget; i++ ) {
      float4 fapprox = tree->bodyAcc[i];
      float4 fdirect = tree->bodyAcc2[i];
      diff1 += (fapprox.w - fdirect.w) * (fapprox.w - fdirect.w);
      diff2 += (fapprox.x - fdirect.x) * (fapprox.x - fdirect.x);
      diff2 += (fapprox.y - fdirect.y) * (fapprox.y - fdirect.y);
      diff2 += (fapprox.z - fdirect.z) * (fapprox.z - fdirect.z);
      norm1 += fdirect.w * fdirect.w;
      norm2 += fdirect.x * fdirect.x;
      norm2 += fdirect.y * fdirect.y;
      norm2 += fdirect.z * fdirect.z;
    }
    printf("Direct: %lf\n",toc-tic);
    printf("P Err : %f\n",sqrtf(diff1/norm1));
    printf("F Err : %f\n",sqrtf(diff2/norm2));
    delete tree;
  }
  return 0;
}
